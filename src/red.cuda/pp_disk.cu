#include "hip/hip_runtime.h"
// includes system
#include <iostream>
#include <iomanip>
#include <string>
#include <fstream>

// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes project
#include "nbody_exception.h"
#include "pp_disk.h"
#include "redutilcu.h"
#include "red_constants.h"
#include "red_macro.h"
#include "red_type.h"

using namespace std;
using namespace redutilcu;

#define GAS_REDUCTION_THRESHOLD 1.0e-6
#define GAS_INNER_EDGE 0.1              // [AU]


__constant__ var_t dc_threshold[THRESHOLD_N];
__constant__ analytic_gas_disk_params_t dc_anal_gd_params;
__constant__ fargo_gas_disk_params_t dc_fargo_gd_params;

///****************** DEVICE functions begins here ******************/

/****************** KERNEL functions begins here ******************/

namespace pp_disk_utility
{
static __host__ __device__ 
	void store_event_data
	(
		event_name_t name,
		ttt_t t,
		var_t d,
		int idx1,
		int idx2,
		const param_t* p,
		const vec_t* r,
		const vec_t* v,
		const body_metadata_t* body_md,
		event_data_t *evnt)
{
	evnt->event_name = name;
	evnt->d = d;
	evnt->t = t;
	evnt->id1 = body_md[idx1].id;
	evnt->id2 = body_md[idx2].id;
	evnt->idx1 = idx1;
	evnt->idx2 = idx2;
	evnt->r1 = r[idx1];
	evnt->v1 = v[idx1];
	evnt->r2 = r[idx2];
	evnt->v2 = v[idx2];

	if (EVENT_NAME_EJECTION == name)
	{
		evnt->p1 = p[idx1];
		evnt->p2 = p[idx2];

		evnt->rs = evnt->r1;
		evnt->vs = evnt->v1;
		evnt->ps = evnt->p1;
	}
}


__host__ __device__
	var_t reduction_factor(gas_decrease_t gas_decrease, ttt_t t0, ttt_t t1, ttt_t e_folding_time, ttt_t t)
{
	switch (gas_decrease) 
	{
	case GAS_DENSITY_CONSTANT:
		return 1.0;
	case GAS_DENSITY_DECREASE_LINEAR:
		if (t <= t0)
		{
			return 1.0;
		}
		else if (t0 < t && t <= t1 && t0 != t1)
		{
			return 1.0 - (t - t0)/(t1 - t0);
		}
		else
		{
			return 0.0;
		}
	case GAS_DENSITY_DECREASE_EXPONENTIAL:
		return exp(-(t - t0)/e_folding_time);
	default:
		return 1.0;
	}
}

__host__ __device__
	var_t get_density(var2_t sch, var2_t rho, const vec_t* rVec)
{
	var_t density = 0.0;

	var_t r		= sqrt(SQR(rVec->x) + SQR(rVec->y));
	var_t h		= sch.x * pow(r, sch.y);
	var_t arg	= SQR(rVec->z/h);
	if (GAS_INNER_EDGE < r)
	{
		density	= rho.x * pow(r, rho.y) * exp(-arg);
	}
	else
	{
		var_t a	= rho.x * pow(GAS_INNER_EDGE, rho.y - 4.0);
		density	= a * SQR(SQR(r)) * exp(-arg);
	}

	return density;
}


__host__ __device__
	vec_t circular_velocity(var_t mu, const vec_t* rVec)
{
	vec_t result = {0.0, 0.0, 0.0, 0.0};

	var_t r  = sqrt(SQR(rVec->x) + SQR(rVec->y));
	var_t vc = sqrt(mu/r);

	var_t p = 0.0;
	if (rVec->x == 0.0 && rVec->y == 0.0)
	{
		return result;
	}
	else if (rVec->y == 0.0)
	{
		result.y = rVec->x > 0.0 ? vc : -vc;
	}
	else if (rVec->x == 0.0)
	{
		result.x = rVec->y > 0.0 ? -vc : vc;
	}
	else if (rVec->x >= rVec->y)
	{
		p = rVec->y / rVec->x;
		result.y = rVec->x >= 0 ? vc/sqrt(1.0 + SQR(p)) : -vc/sqrt(1.0 + SQR(p));
		result.x = -result.y*p;
	}
	else
	{
		p = rVec->x / rVec->y;
		result.x = rVec->y >= 0 ? -vc/sqrt(1.0 + SQR(p)) : vc/sqrt(1.0 + SQR(p));
		result.y = -result.x*p;
	}

	return result;
}

__host__ __device__
	vec_t get_velocity(var_t mu, var2_t eta, const vec_t* rVec)
{
	vec_t v_gas = circular_velocity(mu, rVec);
	var_t r = sqrt(SQR(rVec->x) + SQR(rVec->y));

	var_t v = sqrt(1.0 - 2.0*eta.x * pow(r, eta.y));
	v_gas.x *= v;
	v_gas.y *= v;
	
	return v_gas;
}
} /* pp_disk_utility */

namespace kernel_pp_disk
{
static __global__
	void check_for_ejection_hit_centrum
	(
		ttt_t t, 
		interaction_bound int_bound, 
		const param_t* p, 
		const vec_t* r, 
		const vec_t* v, 
		body_metadata_t* body_md, 
		event_data_t* events,
		int *event_counter
	)
{
	const int i = int_bound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	// Ignore the star, the padding particles (whose id = 0) and the inactive bodies (whose id < 0)
	if (i < int_bound.sink.y && body_md[i].id > 0 && body_md[i].body_type != BODY_TYPE_STAR)
	{
		unsigned int k = 0;

		// Calculate the distance from the barycenter
		var_t r2 = SQR(r[i].x) + SQR(r[i].y) + SQR(r[i].z);
		if (0.0 < dc_threshold[THRESHOLD_EJECTION_DISTANCE] && dc_threshold[THRESHOLD_EJECTION_DISTANCE_SQUARED] < r2)
		{
			k = atomicAdd(event_counter, 1);
			pp_disk_utility::store_event_data(EVENT_NAME_EJECTION, t, sqrt(r2), 0, i, p, r, v, body_md, &events[k]);
			//printf("t = %20.10le d = %20.10le %d. EJECTION detected: id: %5d id: %5d\n", t, sqrt(dVec.w), k+1, body_md[0].id, body_md[i].id);

			//events[k].event_name = EVENT_NAME_EJECTION;
			//events[k].d = sqrt(r2); //sqrt(dVec.w);
			//events[k].t = t;
			//events[k].id1 = body_md[0].id;
			//events[k].id2 = body_md[i].id;
			//events[k].idx1 = 0;
			//events[k].idx2 = i;
			//events[k].r1 = r[0];
			//events[k].v1 = v[0];
			//events[k].r2 = r[i];
			//events[k].v2 = v[i];

			//events[k].p1 = p[0];
			//events[k].p2 = p[i];
			//events[k].ps = p[0];
			//events[k].rs = r[0];
			//events[k].vs = v[0];

			// Make the body inactive
			body_md[i].id *= -1;
		}
		else if (0.0 < dc_threshold[THRESHOLD_HIT_CENTRUM_DISTANCE] && dc_threshold[THRESHOLD_HIT_CENTRUM_DISTANCE_SQUARED] > r2)
		{
			k = atomicAdd(event_counter, 1);
			pp_disk_utility::store_event_data(EVENT_NAME_HIT_CENTRUM, t, sqrt(r2), 0, i, p, r, v, body_md, &events[k]);
			//printf("t = %20.10le d = %20.10le %d. HIT_CENTRUM detected: id: %5d id: %5d\n", t, sqrt(dVec.w), k+1, body_md[0].id, body_md[i].id);

			//events[k].event_name = EVENT_NAME_HIT_CENTRUM;
			//events[k].d = sqrt(r2); //sqrt(dVec.w);
			//events[k].t = t;
			//events[k].id1 = body_md[0].id;
			//events[k].id2 = body_md[i].id;
			//events[k].idx1 = 0;
			//events[k].idx2 = i;
			//events[k].r1 = r[0];
			//events[k].v1 = v[0];
			//events[k].r2 = r[i];
			//events[k].v2 = v[i];
			// Make the body inactive
			body_md[i].id *= -1;
		}
	}
}

static __global__
	void calc_grav_accel_int_mul_of_thread_per_block
	(
		ttt_t t, 
		interaction_bound int_bound, 
		const body_metadata_t* body_md, 
		const param_t* p, 
		const vec_t* r, 
		const vec_t* v, 
		vec_t* a,
		event_data_t* events,
		int *event_counter
	)
{
	const int i = int_bound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	vec_t dVec;
	// This line (beyond my depth) speeds up the kernel
	a[i].x = a[i].y = a[i].z = a[i].w = 0.0;
	for (int j = int_bound.source.x; j < int_bound.source.y; j++) 
	{
		/* Skip the body with the same index */
		if (i == j)
		{
			continue;
		}
		// 3 FLOP
		dVec.x = r[j].x - r[i].x;
		dVec.y = r[j].y - r[i].y;
		dVec.z = r[j].z - r[i].z;
		// 5 FLOP
		dVec.w = SQR(dVec.x) + SQR(dVec.y) + SQR(dVec.z);	// = r2

		// 20 FLOP
		var_t d = sqrt(dVec.w);								// = r
		// 2 FLOP
		dVec.w = p[j].mass / (d*dVec.w);					// = m / r^3
		// 6 FLOP
		a[i].x += dVec.w * dVec.x;
		a[i].y += dVec.w * dVec.y;
		a[i].z += dVec.w * dVec.z;

		if (i > 0 && i > j && d < dc_threshold[THRESHOLD_RADII_ENHANCE_FACTOR] * (p[i].radius + p[j].radius))
		{
			unsigned int k = atomicAdd(event_counter, 1);

			int survivIdx = i;
			int mergerIdx = j;
			if (p[mergerIdx].mass > p[survivIdx].mass)
			{
				int m = survivIdx;
				survivIdx = mergerIdx;
				mergerIdx = m;
			}
			//printf("t = %20.10le d = %20.10le %d. COLLISION detected: id: %5d id: %5d\n", t, d, k+1, body_md[survivIdx].id, body_md[mergerIdx].id);

			pp_disk_utility::store_event_data(EVENT_NAME_COLLISION, t, d, survivIdx, mergerIdx, p, r, v, body_md, &events[k]);

			//events[k].event_name = EVENT_NAME_COLLISION;
			//events[k].d = d;
			//events[k].t = t;
			//events[k].id1 = body_md[survivIdx].id;
			//events[k].id2 = body_md[mergerIdx].id;
			//events[k].idx1 = survivIdx;
			//events[k].idx2 = mergerIdx;
			//events[k].r1 = r[survivIdx];
			//events[k].v1 = v[survivIdx];
			//events[k].r2 = r[mergerIdx];
			//events[k].v2 = v[mergerIdx];
		}
	}
}

static __global__
	void calc_grav_accel
	(
		ttt_t t, 
		interaction_bound int_bound, 
		const body_metadata_t* body_md, 
		const param_t* p, 
		const vec_t* r, 
		const vec_t* v, 
		vec_t* a,
		event_data_t* events,
		int *event_counter
	)
{
	const int i = int_bound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	if (i < int_bound.sink.y)
	{
		a[i].x = a[i].y = a[i].z = a[i].w = 0.0;
		if (0 < body_md[i].id)
		{
			vec_t dVec = {0.0, 0.0, 0.0, 0.0};
			for (int j = int_bound.source.x; j < int_bound.source.y; j++) 
			{
				/* Skip the body with the same index and those which are inactive ie. id < 0 */
				if (i == j || 0 > body_md[j].id)
				{
					continue;
				}
				// 3 FLOP
				dVec.x = r[j].x - r[i].x;
				dVec.y = r[j].y - r[i].y;
				dVec.z = r[j].z - r[i].z;
				// 5 FLOP
				dVec.w = SQR(dVec.x) + SQR(dVec.y) + SQR(dVec.z);	// = r2

				// 20 FLOP
				var_t d = sqrt(dVec.w);								// = r
				// 2 FLOP
				dVec.w = p[j].mass / (d*dVec.w);
				// 6 FLOP
				a[i].x += dVec.w * dVec.x;
				a[i].y += dVec.w * dVec.y;
				a[i].z += dVec.w * dVec.z;

				// Check for collision - ignore the star (i > 0 criterium)
				// The data of the collision will be stored for the body with the greater index (test particles can collide with massive bodies)
				// If i < j is the condition than test particles can not collide with massive bodies
				if (0 < i && i > j && d < dc_threshold[THRESHOLD_RADII_ENHANCE_FACTOR] * (p[i].radius + p[j].radius))
				{
					unsigned int k = atomicAdd(event_counter, 1);

					int survivIdx = i;
					int mergerIdx = j;
					if (p[mergerIdx].mass > p[survivIdx].mass)
					{
						int t = survivIdx;
						survivIdx = mergerIdx;
						mergerIdx = t;
					}
					//printf("t = %20.10le d = %20.10le %d. COLLISION detected: id: %5d id: %5d\n", t, d, k+1, body_md[survivIdx].id, body_md[mergerIdx].id);

					pp_disk_utility::store_event_data(EVENT_NAME_COLLISION, t, d, survivIdx, mergerIdx, p, r, v, body_md, &events[k]);

					//events[k].event_name = EVENT_NAME_COLLISION;
					//events[k].d = d;
					//events[k].t = t;
					//events[k].id1 = body_md[survivIdx].id;
					//events[k].id2 = body_md[mergerIdx].id;
					//events[k].idx1 = survivIdx;
					//events[k].idx2 = mergerIdx;
					//events[k].r1 = r[survivIdx];
					//events[k].v1 = v[survivIdx];
					//events[k].r2 = r[mergerIdx];
					//events[k].v2 = v[mergerIdx];
				}
			} // 36 FLOP
		}
	}
}

static __global__
	void calc_drag_accel_NSI
	(
		ttt_t curr_t,
		interaction_bound int_bound, 
		const body_metadata_t* body_md, 
		const param_t* p, 
		const vec_t* r, 
		const vec_t* v, 
		vec_t* a
	)
{
	const int i = int_bound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	if (i < int_bound.sink.y)
	{
		var_t decr_fact = pp_disk_utility::reduction_factor(dc_anal_gd_params.gas_decrease, dc_anal_gd_params.t0, dc_anal_gd_params.t1, dc_anal_gd_params.e_folding_time, curr_t);
		// TODO: export 1.0e-6 into the gas disk description file
		if (GAS_REDUCTION_THRESHOLD > decr_fact)
		{
			return;
		}

		var_t m_star = p[0].mass;
		var_t mu        = 1.0 * (m_star + p[i].mass);
		vec_t v_g       = pp_disk_utility::get_velocity(mu, dc_anal_gd_params.eta, &r[i]);
		vec_t u         = {v_g.x - v[i].x, v_g.y - v[i].y, v_g.z - v[i].z, 0.0};
		var_t u_n       = sqrt(SQR(u.x) + SQR(u.y) + SQR(u.z));
		var_t density_g = pp_disk_utility::get_density(dc_anal_gd_params.sch, dc_anal_gd_params.rho, &r[i]);

		var_t f = decr_fact * (3.0 * p[i].cd * density_g * u_n) / (8.0 * p[i].radius * p[i].density);

		a[i].x += f * u.x;
		a[i].y += f * u.y;
		a[i].z += f * u.z;
	}
}

} /* kernel_pp_disk */

namespace kernel_utility
{
static __global__
	void print_body_metadata(int n, const body_metadata_t* body_md)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		printf("body_md[%4d].id          : %20.4d\n", i,   body_md[i].id);
		printf("body_md[%4d].body_type   : %20.4d\n", i,   body_md[i].body_type);
		printf("body_md[%4d].mig_type    : %20.4d\n", i,   body_md[i].mig_type);
		printf("body_md[%4d].mig_stop_at : %20.16lf\n", i, body_md[i].mig_stop_at);
	}
}

static __global__
	void print_epochs(int n, const ttt_t* epoch)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		printf("epoch[%4d] : %20.16lf\n", i, epoch[i]);
	}
}

static __global__
	void print_vector(int n, const vec_t* v)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n)
	{
		printf("[%d]: (%20.16lf, %20.16lf, %20.16lf, %20.16lf)\n", tid, v[tid].x, v[tid].y, v[tid].z, v[tid].w);
	}
}

static __global__
	void print_constant_memory()
{
	printf("dc_threshold[THRESHOLD_HIT_CENTRUM_DISTANCE        ] : %lf\n", dc_threshold[THRESHOLD_HIT_CENTRUM_DISTANCE]);
	printf("dc_threshold[THRESHOLD_EJECTION_DISTANCE           ] : %lf\n", dc_threshold[THRESHOLD_EJECTION_DISTANCE]);
	printf("dc_threshold[THRESHOLD_RADII_ENHANCE_FACTOR        ] : %lf\n", dc_threshold[THRESHOLD_RADII_ENHANCE_FACTOR]);
	printf("dc_threshold[THRESHOLD_HIT_CENTRUM_DISTANCE_SQUARED] : %lf\n", dc_threshold[THRESHOLD_HIT_CENTRUM_DISTANCE_SQUARED]);
	printf("dc_threshold[THRESHOLD_EJECTION_DISTANCE_SQUARED   ] : %lf\n", dc_threshold[THRESHOLD_EJECTION_DISTANCE_SQUARED]);
}
} /* kernel_utility */


void pp_disk::test_call_kernel_print_sim_data()
{
	const int n_total = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	set_kernel_launch_param(n_total);

	kernel_utility::print_vector<<<grid, block>>>(n_total, sim_data->d_y[0]);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		throw nbody_exception("kernel_utility::print_vector failed", cudaStatus);
	}
	hipDeviceSynchronize();

	kernel_utility::print_vector<<<grid, block>>>(n_total, sim_data->d_y[1]);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		throw nbody_exception("kernel_utility::print_vector failed", cudaStatus);
	}
	hipDeviceSynchronize();

	kernel_utility::print_vector<<<grid, block>>>(n_total, (vec_t*)sim_data->d_p);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		throw nbody_exception("kernel_utility::print_vector failed", cudaStatus);
	}
	hipDeviceSynchronize();

	kernel_utility::print_body_metadata<<<grid, block>>>(n_total, sim_data->d_body_md);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		throw nbody_exception("kernel_utility::print_body_metadata failed", cudaStatus);
	}
	hipDeviceSynchronize();

	kernel_utility::print_epochs<<<grid, block>>>(n_total, sim_data->d_epoch);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		throw nbody_exception("kernel_utility::print_epochs failed", cudaStatus);
	}
	hipDeviceSynchronize();

	kernel_utility::print_constant_memory<<<1, 1>>>();
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		throw nbody_exception("kernel_utility::print_constant_memory failed", cudaStatus);
	}
	hipDeviceSynchronize();
}

void pp_disk::set_kernel_launch_param(int n_data)
{
	int n_thread = min(n_tpb, n_data);
	int n_block = (n_data + n_thread - 1)/n_thread;

	grid.x	= n_block;
	block.x = n_thread;
}

void pp_disk::cpu_calc_drag_accel(ttt_t curr_t, const vec_t* r, const vec_t* v, vec_t* dy)
{
	int n_sink = n_bodies->get_n_NSI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_GD();
		cpu_calc_drag_accel_NSI(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy);
	}
}


void pp_disk::cpu_calc_drag_accel_NSI(ttt_t curr_t, interaction_bound int_bound, const body_metadata_t* body_md, const param_t* p, const vec_t* r, const vec_t* v, vec_t* a)
{
	var_t decr_fact = pp_disk_utility::reduction_factor(a_gd->params.gas_decrease, a_gd->params.t0, a_gd->params.t1, a_gd->params.e_folding_time, curr_t);
	// TODO: export 1.0e-6 into the gas disk description file
	if (GAS_REDUCTION_THRESHOLD > decr_fact)
	{
		return;
	}

	var_t m_star = p[0].mass;
	for (int i = int_bound.sink.x; i < int_bound.sink.y; i++)
	{
		var_t mu        = 1.0 * (m_star + p[i].mass);
		vec_t v_g       = pp_disk_utility::get_velocity(mu, a_gd->params.eta, &r[i]);
		vec_t u         = {v_g.x - v[i].x, v_g.y - v[i].y, v_g.z - v[i].z, 0.0};
		var_t u_n       = sqrt(SQR(u.x) + SQR(u.y) + SQR(u.z));
		var_t density_g = pp_disk_utility::get_density(a_gd->params.sch, a_gd->params.rho, &r[i]);

		var_t f = decr_fact * (3.0 * p[i].cd * density_g * u_n) / (8.0 * p[i].radius * p[i].density);

		a[i].x += f * u.x;
		a[i].y += f * u.y;
		a[i].z += f * u.z;

		//var_t rhoGas = rFactor * gas_density_at(gasDisk, (vec_t*)&coor[bodyIdx]);
		//var_t r = norm((vec_t*)&coor[bodyIdx]);

		//vec_t u;
		//u.x	= velo[bodyIdx].x - vGas.x;
		//u.y	= velo[bodyIdx].y - vGas.y;
		//u.z	= velo[bodyIdx].z - vGas.z;
		//var_t C	= 0.0;

		//var_t lambda = gasDisk->mfp.x * pow(r, gasDisk->mfp.y);
		//// Epstein-regime:
		//if (     params[bodyIdx].radius <= 0.1 * lambda)
		//{
		//	var_t vth = mean_thermal_speed_CMU(gasDisk, r);
		//	C = params[bodyIdx].gamma_epstein * vth * rhoGas;
		//}
		//// Stokes-regime:
		//else if (params[bodyIdx].radius >= 10.0 * lambda)
		//{
		//	C = params[bodyIdx].gamma_stokes * norm(&u) * rhoGas;
		//}
		//// Transition-regime:
		//else
		//{

		//}

		//acce[tid].x = -C * u.x;
		//acce[tid].y = -C * u.y;
		//acce[tid].z = -C * u.z;
		//acce[tid].w = 0.0;

		//printf("acce[tid].x: %10le\n", acce[tid].x);
		//printf("acce[tid].y: %10le\n", acce[tid].y);
		//printf("acce[tid].z: %10le\n", acce[tid].z);
	}
}

void pp_disk::cpu_calc_grav_accel_SI(ttt_t curr_t, interaction_bound int_bound, const body_metadata_t* body_md, const param_t* p, const vec_t* r, const vec_t* v, vec_t* a, event_data_t* events, int *event_counter)
{
	for (int i = int_bound.sink.x; i < int_bound.sink.y; i++)
	{
		a[i].x = a[i].y = a[i].z = a[i].w = 0.0;
		if (0 < body_md[i].id)
		{
			vec_t dVec = {0.0, 0.0, 0.0, 0.0};
			for (int j = int_bound.source.x; j < int_bound.source.y; j++) 
			{
				/* Skip the body with the same index and those which are inactive ie. id < 0 */
				if (i == j || 0 > body_md[j].id)
				{
					continue;
				}
				// 3 FLOP
				dVec.x = r[j].x - r[i].x;
				dVec.y = r[j].y - r[i].y;
				dVec.z = r[j].z - r[i].z;
				// 5 FLOP
				dVec.w = SQR(dVec.x) + SQR(dVec.y) + SQR(dVec.z);	// = r2

				// 20 FLOP
				var_t d = sqrt(dVec.w);								// = r
				// 2 FLOP
				dVec.w = p[j].mass / (d*dVec.w);
				// 6 FLOP
				a[i].x += dVec.w * dVec.x;
				a[i].y += dVec.w * dVec.y;
				a[i].z += dVec.w * dVec.z;

				// Check for collision - ignore the star (i > 0 criterium)
				// The data of the collision will be stored for the body with the greater index (test particles can collide with massive bodies)
				// If i < j is the condition than test particles can not collide with massive bodies
				if (i > 0 && i > j && d < threshold[THRESHOLD_RADII_ENHANCE_FACTOR] * (p[i].radius + p[j].radius))
				{
					int k = *event_counter;

					int survivIdx = i;
					int mergerIdx = j;
					if (p[mergerIdx].mass > p[survivIdx].mass)
					{
						int t = survivIdx;
						survivIdx = mergerIdx;
						mergerIdx = t;
					}
					printf("t = %20.10le d = %20.10le %d. COLLISION detected: id: %5d id: %5d\n", t, d, k+1, body_md[survivIdx].id, body_md[mergerIdx].id);

					events[k].event_name = EVENT_NAME_COLLISION;
					events[k].d = d;
					events[k].t = t;
					events[k].id1 = body_md[survivIdx].id;
					events[k].id2 = body_md[mergerIdx].id;
					events[k].idx1 = survivIdx;
					events[k].idx2 = mergerIdx;
					events[k].r1 = r[survivIdx];
					events[k].v1 = v[survivIdx];
					events[k].r2 = r[mergerIdx];
					events[k].v2 = v[mergerIdx];

					(*event_counter)++;
				}
			} // 36 FLOP
		}
	}
}

void pp_disk::cpu_calc_grav_accel_NI( ttt_t curr_t, interaction_bound int_bound, const body_metadata_t* body_md, const param_t* p, const vec_t* r, const vec_t* v, vec_t* a, event_data_t* events, int *event_counter)
{
	cpu_calc_grav_accel_SI(t, int_bound, body_md, p, r, v, a, events, event_counter);
}

void pp_disk::cpu_calc_grav_accel_NSI(ttt_t curr_t, interaction_bound int_bound, const body_metadata_t* body_md, const param_t* p, const vec_t* r, const vec_t* v, vec_t* a, event_data_t* events, int *event_counter)
{
	cpu_calc_grav_accel_SI(t, int_bound, body_md, p, r, v, a, events, event_counter);
}

void pp_disk::cpu_calc_grav_accel(ttt_t curr_t, const vec_t* r, const vec_t* v, vec_t* dy)
{
	int n_sink = n_bodies->get_n_SI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_SI();
		cpu_calc_grav_accel_SI(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, events, &event_counter);
	}

	n_sink = n_bodies->get_n_NSI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_NSI();
		cpu_calc_grav_accel_NSI(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, events, &event_counter);
	}

	n_sink = n_bodies->get_n_NI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_NI();
		cpu_calc_grav_accel_NI(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, events, &event_counter);
	}
}

void pp_disk::call_kernel_calc_grav_accel(ttt_t curr_t, const vec_t* r, const vec_t* v, vec_t* dy)
{
	hipError_t cudaStatus = hipSuccess;
	
	int n_sink = use_padded_storage ? n_bodies->get_n_prime_SI() : n_bodies->get_n_SI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_SI();
		set_kernel_launch_param(n_sink);

		if (use_padded_storage)
		{
			kernel_pp_disk::calc_grav_accel_int_mul_of_thread_per_block<<<grid, block>>>
				(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, d_events, d_event_counter);
		}
		else
		{
			kernel_pp_disk::calc_grav_accel<<<grid, block>>>
				(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, d_events, d_event_counter);
		}
		cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus)
		{
			throw nbody_exception("kernel_pp_disk::calc_grav_accel failed", cudaStatus);
		}
	}

	n_sink = use_padded_storage ? n_bodies->get_n_prime_NSI() : n_bodies->get_n_NSI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_NSI();
		set_kernel_launch_param(n_sink);

		if (use_padded_storage)
		{
			kernel_pp_disk::calc_grav_accel_int_mul_of_thread_per_block<<<grid, block>>>
				(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, d_events, d_event_counter);
		}
		else
		{
			kernel_pp_disk::calc_grav_accel<<<grid, block>>>
				(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, d_events, d_event_counter);
		}
		cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus)
		{
			throw nbody_exception("kernel_pp_disk::calc_grav_accel failed", cudaStatus);
		}
	}

	n_sink = use_padded_storage ? n_bodies->get_n_prime_NI() : n_bodies->get_n_NI();
	if (0 < n_sink)
	{
		interaction_bound int_bound = n_bodies->get_bound_NI();
		set_kernel_launch_param(n_sink);

		if (use_padded_storage)
		{
			kernel_pp_disk::calc_grav_accel_int_mul_of_thread_per_block<<<grid, block>>>
				(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, d_events, d_event_counter);
		}
		else
		{
			kernel_pp_disk::calc_grav_accel<<<grid, block>>>
				(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy, d_events, d_event_counter);
		}
		cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus)
		{
			throw nbody_exception("kernel_pp_disk::calc_grav_accel failed", cudaStatus);
		}
	}
}

void pp_disk::call_kernel_calc_drag_accel(ttt_t curr_t, const vec_t* r, const vec_t* v, vec_t* dy)
{
	int n_sink = n_bodies->get_n_NSI();
	if (0 < n_sink)
	{
		set_kernel_launch_param(n_sink);
		interaction_bound int_bound = n_bodies->get_bound_GD();
		kernel_pp_disk::calc_drag_accel_NSI<<<grid, block>>>(curr_t, int_bound, sim_data->body_md, sim_data->p, r, v, dy);
	}
}

bool pp_disk::check_for_ejection_hit_centrum()
{
	// Number of ejection + hit centrum events
	int n_event = 0;
	switch (comp_dev)
	{
	case COMPUTING_DEVICE_CPU:
		n_event = cpu_check_for_ejection_hit_centrum();
		break;
	case COMPUTING_DEVICE_GPU:
		n_event = call_kernel_check_for_ejection_hit_centrum();
		break;
	}

	if (0 < n_event)
	{
		if (COMPUTING_DEVICE_GPU == comp_dev)
		{
			copy_event_data_to_host();
		}
		// handle_ejection_hit_centrum() will create sp_events vector which will explicitly written to the disk via print_event_data()
		handle_ejection_hit_centrum();
		cout << n_ejection[   EVENT_COUNTER_NAME_LAST_STEP] << " ejection ";
		cout << n_hit_centrum[EVENT_COUNTER_NAME_LAST_STEP] << " hit_centrum event(s) occured" << endl;

		n_ejection[   EVENT_COUNTER_NAME_LAST_STEP] = 0;
		n_hit_centrum[EVENT_COUNTER_NAME_LAST_STEP] = 0;

		return true;
	}

	return false;
}

bool pp_disk::check_for_collision()
{
	// Number of collision
	int n_event = get_n_event();

	if (0 < n_event)
	{
		if (COMPUTING_DEVICE_GPU == comp_dev)
		{
			copy_event_data_to_host();
		}
		// handle_collision() will create sp_events vector which will explicitly written to the disk via print_event_data()
		handle_collision();
		cout << n_collision[EVENT_COUNTER_NAME_LAST_STEP] << " collision event(s) occurred" << endl;

		n_collision[EVENT_COUNTER_NAME_LAST_STEP] = 0;

		return true;
	}

	return false;
}

bool pp_disk::check_for_rebuild_vectors(int n)
{
	if (n_event[EVENT_COUNTER_NAME_LAST_CLEAR] >= n)
	{
		if (COMPUTING_DEVICE_GPU == comp_dev)
		{
			copy_to_host();
		}
		// Rebuild the vectors and remove inactive bodies
		remove_inactive_bodies();
		set_event_counter(EVENT_COUNTER_NAME_LAST_CLEAR, 0);
		return true;
	}
	return false;
}

void pp_disk::store_event_data(event_name_t name, ttt_t t, var_t d, int idx1, int idx2, event_data_t *evnt)
{
	evnt->event_name = name;
	evnt->d = d;
	evnt->t = t;
	evnt->id1 = sim_data->body_md[idx1].id;
	evnt->id2 = sim_data->body_md[idx2].id;
	evnt->idx1 = idx1;
	evnt->idx2 = idx2;
	evnt->r1 = sim_data->y[0][idx1];
	evnt->v1 = sim_data->y[1][idx1];
	evnt->r2 = sim_data->y[0][idx2];
	evnt->v2 = sim_data->y[1][idx2];

	if (EVENT_NAME_EJECTION == name)
	{
		evnt->p1 = sim_data->h_p[idx1];
		evnt->p2 = sim_data->h_p[idx2];

		evnt->rs = evnt->r1;
		evnt->vs = evnt->v1;
		evnt->ps = evnt->p1;
	}
}

int pp_disk::cpu_check_for_ejection_hit_centrum()
{
	const vec_t* r = sim_data->y[0];
	const vec_t* v = sim_data->y[1];
	body_metadata_t* body_md = sim_data->body_md;
	
	int n_total = n_bodies->get_n_total();
	interaction_bound int_bound(0, n_total, 0, 0);

	for (int i = int_bound.sink.x; i < int_bound.sink.y; i++)
	{
		// Ignore the star and the inactive bodies (whose id < 0)
		if (0 < sim_data->body_md[i].id && BODY_TYPE_STAR != sim_data->body_md[i].body_type)
		{
			//int k = 0;

			// Calculate the distance from the barycenter
			var_t r2 = SQR(r[i].x) + SQR(r[i].y) + SQR(r[i].z);
			if (0.0 < threshold[THRESHOLD_EJECTION_DISTANCE] && threshold[THRESHOLD_EJECTION_DISTANCE_SQUARED] < r2)
			{
				//k = event_counter;
				//printf("t = %20.10le d = %20.10le %d. EJECTION detected: id: %5d id: %5d\n", t, sqrt(dVec.w), k+1, body_md[0].id, body_md[i].id);
				store_event_data(EVENT_NAME_EJECTION, t, sqrt(r2), 0, i, &events[event_counter]);

				// Make the body inactive
				body_md[i].id *= -1;
				event_counter++;
			}
			else if (0.0 < threshold[THRESHOLD_HIT_CENTRUM_DISTANCE] && threshold[THRESHOLD_HIT_CENTRUM_DISTANCE_SQUARED] > r2)
			{
				//k = event_counter;
				//printf("t = %20.10le d = %20.10le %d. HIT_CENTRUM detected: id: %5d id: %5d\n", t, sqrt(dVec.w), k+1, body_md[0].id, body_md[i].id);
				store_event_data(EVENT_NAME_HIT_CENTRUM, t, sqrt(r2), 0, i, &events[event_counter]);

				// Make the body inactive
				body_md[i].id *= -1;
				event_counter++;
			}
		}
	}

	return event_counter;
}

int pp_disk::call_kernel_check_for_ejection_hit_centrum()
{
	hipError_t cudaStatus = hipSuccess;
	
	int n_total = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();
	interaction_bound int_bound(0, n_total, 0, 0);
	set_kernel_launch_param(n_total);

	kernel_pp_disk::check_for_ejection_hit_centrum<<<grid, block>>>
		(t, int_bound, sim_data->p, sim_data->y[0], sim_data->y[1], sim_data->body_md, d_events, d_event_counter);

	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw nbody_exception("kernel_pp_disk::check_for_ejection_hit_centrum failed", cudaStatus);
	}

	return get_n_event();
}

void pp_disk::calc_dydx(int i, int rr, ttt_t curr_t, const vec_t* r, const vec_t* v, vec_t* dy)
{
	hipError_t cudaStatus = hipSuccess;

	const int n_total = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	switch (i)
	{
	case 0:
		if (COMPUTING_DEVICE_CPU == comp_dev)
		{
			memcpy(dy, v, n_total * sizeof(vec_t));
		}
		else
		{
			hipMemcpy(dy, v, n_total * sizeof(vec_t), hipMemcpyDeviceToDevice);
			cudaStatus = HANDLE_ERROR(hipGetLastError());
			if (hipSuccess != cudaStatus)
			{
				throw nbody_exception("hipMemcpy failed", cudaStatus);
			}
		}
		break;
	case 1:  // Calculate accelerations originated from the gravitational force, drag force etc.
		if (COMPUTING_DEVICE_CPU == comp_dev)
		{
			/*
			 * SORREND:
			 * 1. Gravity
			 * 2. other forces
			 */
			cpu_calc_grav_accel(curr_t, r, v, dy);
			// This if will be used to speed-up the calculation when gas drag is also acting on the bodies.
			// (BUT early optimization is the root of much evil)
			if (rr == 0)
			{
			}
			if (GAS_DISK_MODEL_NONE != g_disk_model)
			{
				cpu_calc_drag_accel(curr_t, r, v, dy);
			}
		}
		else
		{
			call_kernel_calc_grav_accel(curr_t, r, v, dy);
			if (rr == 0)
			{
			}
			if (GAS_DISK_MODEL_NONE != g_disk_model)
			{
				call_kernel_calc_drag_accel(curr_t, r, v, dy);
			}
	// DEBUG CODE
	//		hipDeviceSynchronize();
	// END DEBUG CODE
		}
		break;
	}
}

void pp_disk::swap()
{
	for (int i = 0; i < 2; i++)
	{
		::swap(sim_data->yout[i], sim_data->y[i]);
	}
}

void pp_disk::increment_event_counter(int *event_counter)
{
	for (int i = 0; i < EVENT_COUNTER_NAME_N; i++)
	{
		event_counter[i]++;
		// Increment the total number of events
		n_event[i]++;
	}
}

void pp_disk::set_event_counter(event_counter_name_t field, int value)
{
	n_hit_centrum[field] = value;
	n_ejection[field]    = value;
	n_collision[field]   = value;
	n_event[field]       = value;
}

void pp_disk::handle_collision()
{
	create_sp_events();

	// TODO: implement collision graph: bredth-first search
	for (unsigned int i = 0; i < sp_events.size(); i++)
	{
		handle_collision_pair(i, &sp_events[i]);
		increment_event_counter(n_collision);
	}
}

void pp_disk::handle_ejection_hit_centrum()
{
	sp_events.resize(event_counter);

	for (int i = 0; i < event_counter; i++)
	{
		// The events must be copied into sp_events since the print_event_data() write the content of the sp_events to disk.
		sp_events[i] = events[i];
		if (sp_events[i].event_name == EVENT_NAME_EJECTION)
		{
			increment_event_counter(n_ejection);
		}
		else
		{
			handle_collision_pair(i, &sp_events[i]);
			increment_event_counter(n_hit_centrum);
		}
	}
}

void pp_disk::create_sp_events()
{
	sp_events.resize(event_counter);

	bool *processed = new bool[event_counter];
	for (int i = 0; i < event_counter; i++)
	{
		processed[i] = false;
	}

	int n = 0;
	for (int k = 0; k < event_counter; k++)
	{
		if (processed[k] == false)
		{
			processed[k] = true;
			sp_events[n] = events[k];
		}
		else
		{
			continue;
		}
		for (int i = k + 1; i < event_counter; i++)
		{
			if (sp_events[n].id1 == events[i].id1 && sp_events[n].id2 == events[i].id2)
			{
				processed[i] = true;
				if (sp_events[n].t > events[i].t)
				{
					sp_events[n] = events[i];
				}
			}
		}
		n++;
	}
	delete[] processed;

	sp_events.resize(n);
}

void pp_disk::handle_collision_pair(int i, event_data_t *collision)
{
	int survivIdx = collision->idx1;
	int mergerIdx = collision->idx2;

	if (BODY_TYPE_SUPERPLANETESIMAL == sim_data->h_body_md[mergerIdx].body_type)
	{
		// TODO: implement collision between a body and a super-planetesimal
		throw string("Collision between a massive body and a super-planetesimal is not yet implemented.");
	}

	collision->p1 = sim_data->h_p[survivIdx];
	collision->p2 = sim_data->h_p[mergerIdx];

	// Calculate position and velocitiy of the new object
	tools::calc_position_after_collision(collision->p1.mass, collision->p2.mass, &(collision->r1), &(collision->r2), collision->rs);
	tools::calc_velocity_after_collision(collision->p1.mass, collision->p2.mass, &(collision->v1), &(collision->v2), collision->vs);
	// Update position and velocity of survivor
	sim_data->h_y[0][survivIdx] = collision->rs;
	sim_data->h_y[1][survivIdx] = collision->vs;

	// Calculate physical properties of the new object
	tools::calc_physical_properties(collision->p1, collision->p2, collision->ps);
	// Update physical properties of survivor
	sim_data->h_p[survivIdx] = collision->ps;

	// Make the merged body inactive 
	sim_data->h_body_md[mergerIdx].id *= -1;
	// Set its parameters to zero
	sim_data->h_p[mergerIdx].mass    = 0.0;
	sim_data->h_p[mergerIdx].density = 0.0;
	sim_data->h_p[mergerIdx].radius  = 0.0;
	// and push it radialy extremly far away with zero velocity
	sim_data->h_y[0][mergerIdx].x = 1.0e20;
	sim_data->h_y[0][mergerIdx].y = 0.0;
	sim_data->h_y[0][mergerIdx].z = 0.0;
	sim_data->h_y[1][mergerIdx].x = 0.0;
	sim_data->h_y[1][mergerIdx].y = 0.0;
	sim_data->h_y[1][mergerIdx].z = 0.0;

	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		copy_vector_to_device((void **)&sim_data->d_y[0][survivIdx],	(void *)&sim_data->h_y[0][survivIdx],	 sizeof(vec_t));
		copy_vector_to_device((void **)&sim_data->d_y[1][survivIdx],	(void *)&sim_data->h_y[1][survivIdx],	 sizeof(vec_t));
		copy_vector_to_device((void **)&sim_data->d_p[survivIdx],		(void *)&sim_data->h_p[survivIdx],       sizeof(param_t));

		copy_vector_to_device((void **)&sim_data->d_y[0][mergerIdx],	(void *)&sim_data->h_y[0][mergerIdx],    sizeof(vec_t));
		copy_vector_to_device((void **)&sim_data->d_y[1][mergerIdx],	(void *)&sim_data->h_y[1][mergerIdx],    sizeof(vec_t));
		copy_vector_to_device((void **)&sim_data->d_p[mergerIdx],		(void *)&sim_data->h_p[mergerIdx],       sizeof(param_t));
		copy_vector_to_device((void **)&sim_data->d_body_md[mergerIdx],	(void *)&sim_data->h_body_md[mergerIdx], sizeof(body_metadata_t));
	}
}

pp_disk::pp_disk(string& path, int n_tpb, bool use_padded_storage, gas_disk_model_t g_disk_model, computing_device_t comp_dev) :
	n_tpb(n_tpb),
	use_padded_storage(use_padded_storage),
	g_disk_model(g_disk_model),
	comp_dev(comp_dev)
{
	initialize();
	n_bodies = get_number_of_bodies(path);
	allocate_storage();
	redutilcu::create_aliases(comp_dev, sim_data);
	load(path);
}

pp_disk::~pp_disk()
{
	deallocate_host_storage(sim_data);
	FREE_HOST_VECTOR((void **)&events);

	deallocate_device_storage(sim_data);
	FREE_DEVICE_VECTOR((void **)&d_events);
	FREE_DEVICE_VECTOR((void **)&d_event_counter);
	delete sim_data;

	//FREE_HOST_VECTOR(  (void **)&g_disk);
	//FREE_DEVICE_VECTOR((void **)&d_g_disk);
}

void pp_disk::initialize()
{
	t               = 0.0;
	sim_data        = 0x0;
	n_bodies        = 0x0;
	event_counter   = 0;
	d_event_counter = 0x0;
	events          = 0x0;
	d_events        = 0x0;

	for (int i = 0; i < EVENT_COUNTER_NAME_N; i++)
	{
		n_hit_centrum[i] = 0;
		n_ejection[i]    = 0;
		n_collision[i]   = 0;
		n_event[i]       = 0;
	}
}

void pp_disk::allocate_storage()
{
	int n_total = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	sim_data = new sim_data_t;
	
	// These will be only aliases to the actual storage space either in the HOST or DEVICE memory
	sim_data->y.resize(2);
	sim_data->yout.resize(2);

	allocate_host_storage(sim_data, n_total);
	ALLOCATE_HOST_VECTOR((void **)&events, n_total*sizeof(event_data_t));

	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		allocate_device_storage(sim_data, n_total);
		ALLOCATE_DEVICE_VECTOR((void **)&d_events,        n_total*sizeof(event_data_t));
		ALLOCATE_DEVICE_VECTOR((void **)&d_event_counter,       1*sizeof(int));
	}
}

void pp_disk::set_computing_device(computing_device_t device)
{
	// If the execution is already on the requested device than nothing to do
	if (this->comp_dev == device)
	{
		return;
	}

	int n_total = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	switch (device)
	{
	case COMPUTING_DEVICE_CPU:
		copy_to_host();
		clear_event_counter();
		deallocate_device_storage(sim_data);
		FREE_DEVICE_VECTOR((void **)&d_events);
		FREE_DEVICE_VECTOR((void **)&d_event_counter);
		break;
	case COMPUTING_DEVICE_GPU:
		allocate_device_storage(sim_data, n_total);
		ALLOCATE_DEVICE_VECTOR((void **)&d_events,        n_total*sizeof(event_data_t));
		ALLOCATE_DEVICE_VECTOR((void **)&d_event_counter,       1*sizeof(int));

		copy_to_device();
		copy_disk_params_to_device();
		copy_constant_to_device(dc_threshold, this->threshold, THRESHOLD_N*sizeof(var_t));
		copy_vector_to_device((void *)d_event_counter, (void *)&event_counter, 1*sizeof(int));
		break;
	default:
		throw string ("Invalid parameter: computing device was out of range.");
	}
	redutilcu::create_aliases(device, sim_data);

	this->comp_dev = device;
}

number_of_bodies* pp_disk::get_number_of_bodies(string& path)
{
	int ns, ngp, nrp, npp, nspl, npl, ntp;

	ifstream input(path.c_str());
	if (input) 
	{
		input >> ns >> ngp >> nrp >> npp >> nspl >> npl >> ntp;
		input.close();
	}
	else 
	{
		throw string("Cannot open " + path + ".");
	}
    return new number_of_bodies(ns, ngp, nrp, npp, nspl, npl, ntp, n_tpb, use_padded_storage);
}

void pp_disk::remove_inactive_bodies()
{
	int old_n_total = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();
	// Update the numbers after counting the eliminated bodies
	n_bodies->update_numbers(sim_data->h_body_md);

	sim_data_t *sim_data_temp = new sim_data_t;
	// Only the data of the active bodies will be temporarily stored
	allocate_host_storage(sim_data_temp, n_bodies->get_n_total());

	// Copy the data of active bodies to sim_data_temp
	int i = 0;
	int k = 0;
	for ( ; i < old_n_total; i++)
	{
		if (0 < sim_data->h_body_md[i].id && BODY_TYPE_PADDINGPARTICLE > sim_data->h_body_md[i].body_type)
		{
			sim_data_temp->h_y[0][k]    = sim_data->h_y[0][i];
			sim_data_temp->h_y[1][k]    = sim_data->h_y[1][i];
			sim_data_temp->h_p[k]       = sim_data->h_p[i];
			sim_data_temp->h_body_md[k] = sim_data->h_body_md[i];
			k++;
		}
	}
	if (n_bodies->get_n_total() != k)
	{
		throw string("Error: number of copied bodies does not equal to the number of active bodies.");
	}

	int n_SI		= n_bodies->get_n_SI();
	int n_NSI		= n_bodies->get_n_NSI();
	int n_total		= n_bodies->get_n_total();
	int n_prime_SI	= n_bodies->get_n_prime_SI();
	int n_prime_NSI	= n_bodies->get_n_prime_NSI();
	int n_prime_total=n_bodies->get_n_prime_total();

	k = 0;
	i = 0;
	for ( ; i < n_SI; i++, k++)
	{
		sim_data->h_y[0][k]    = sim_data_temp->h_y[0][i];
		sim_data->h_y[1][k]    = sim_data_temp->h_y[1][i];
		sim_data->h_p[k]       = sim_data_temp->h_p[i];
		sim_data->h_body_md[k] = sim_data_temp->h_body_md[i];
	}
    while (use_padded_storage && k < n_prime_SI)
    {
		create_padding_particle(k, sim_data->h_epoch, sim_data->h_body_md, sim_data->h_p, sim_data->h_y[0], sim_data->h_y[1]);
        k++;
    }

	for ( ; i < n_SI + n_NSI; i++, k++)
	{
		sim_data->h_y[0][k]    = sim_data_temp->h_y[0][i];
		sim_data->h_y[1][k]    = sim_data_temp->h_y[1][i];
		sim_data->h_p[k]       = sim_data_temp->h_p[i];
		sim_data->h_body_md[k] = sim_data_temp->h_body_md[i];
	}
    while (use_padded_storage && k < n_prime_SI + n_prime_NSI)
    {
		create_padding_particle(k, sim_data->h_epoch, sim_data->h_body_md, sim_data->h_p, sim_data->h_y[0], sim_data->h_y[1]);
        k++;
    }

	for ( ; i < n_total; i++, k++)
	{
		sim_data->h_y[0][k]    = sim_data_temp->h_y[0][i];
		sim_data->h_y[1][k]    = sim_data_temp->h_y[1][i];
		sim_data->h_p[k]       = sim_data_temp->h_p[i];
		sim_data->h_body_md[k] = sim_data_temp->h_body_md[i];
	}
    while (use_padded_storage && k < n_prime_total)
    {
		create_padding_particle(k, sim_data->h_epoch, sim_data->h_body_md, sim_data->h_p, sim_data->h_y[0], sim_data->h_y[1]);
        k++;
    }

	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		// Copy the active bodies to the device
		copy_to_device();
	}
	deallocate_host_storage(sim_data_temp);

	delete sim_data_temp;
}

void pp_disk::copy_to_device()
{
	int n_body = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	for (int i = 0; i < 2; i++)
	{
		copy_vector_to_device((void *)sim_data->d_y[i],	(void *)sim_data->h_y[i],	 n_body*sizeof(vec_t));
	}
	copy_vector_to_device((void *)sim_data->d_p,		(void *)sim_data->h_p,		 n_body*sizeof(param_t));
	copy_vector_to_device((void *)sim_data->d_body_md,	(void *)sim_data->h_body_md, n_body*sizeof(body_metadata_t));
	copy_vector_to_device((void *)sim_data->d_epoch,	(void *)sim_data->h_epoch,	 n_body*sizeof(ttt_t));

	copy_vector_to_device((void *)d_event_counter,		(void *)&event_counter,		      1*sizeof(int));
}

void pp_disk::copy_to_host()
{
	int n_body = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	for (int i = 0; i < 2; i++)
	{
		copy_vector_to_host((void *)sim_data->h_y[i],	(void *)sim_data->d_y[i],	 n_body*sizeof(vec_t));
	}
	copy_vector_to_host((void *)sim_data->h_p,			(void *)sim_data->d_p,		 n_body*sizeof(param_t));
	copy_vector_to_host((void *)sim_data->h_body_md,	(void *)sim_data->d_body_md, n_body*sizeof(body_metadata_t));
	copy_vector_to_host((void *)sim_data->h_epoch,		(void *)sim_data->d_epoch,	 n_body*sizeof(ttt_t));

	copy_vector_to_host((void *)&event_counter,			(void *)d_event_counter,	      1*sizeof(int));
}

void pp_disk::copy_threshold(const var_t* thrshld)
{
	memcpy(threshold, thrshld, THRESHOLD_N * sizeof(var_t));

	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		copy_constant_to_device(dc_threshold, thrshld, THRESHOLD_N*sizeof(var_t));
	}
}

void pp_disk::copy_disk_params_to_device()
{
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		switch (g_disk_model)
		{
		case GAS_DISK_MODEL_NONE:
			break;
		case GAS_DISK_MODEL_ANALYTIC:
			copy_constant_to_device((void*)&dc_anal_gd_params,  (void*)&(this->a_gd->params), sizeof(analytic_gas_disk_params_t));
			break;
		case GAS_DISK_MODEL_FARGO:
			copy_constant_to_device((void*)&dc_fargo_gd_params, (void*)&(this->f_gd->params), sizeof(fargo_gas_disk_params_t));
			break;
		}
	}
}

void pp_disk::copy_event_data_to_host()
{
	copy_vector_to_host((void *)events, (void *)d_events, event_counter*sizeof(event_data_t));
}

int pp_disk::get_n_event()
{
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		copy_vector_to_host((void *)&event_counter, (void *)d_event_counter, 1*sizeof(int));
	}

	return event_counter;
}

void pp_disk::clear_event_counter()
{
	event_counter = 0;
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		copy_vector_to_device((void *)d_event_counter, (void *)&event_counter, 1*sizeof(int));
	}
}

int pp_disk::get_n_total_event()
{
	return (n_collision[EVENT_COUNTER_NAME_TOTAL] + n_ejection[EVENT_COUNTER_NAME_TOTAL] + n_hit_centrum[EVENT_COUNTER_NAME_TOTAL]);
}

var_t pp_disk::get_mass_of_star()
{
	int n = use_padded_storage ? n_bodies->get_n_prime_massive() : n_bodies->get_n_massive();

	body_metadata_t* body_md = sim_data->h_body_md;
	for (int j = 0; j < n; j++ )
	{
		if (body_md[j].body_type == BODY_TYPE_STAR)
		{
			return sim_data->h_p[j].mass;
		}
	}
	throw string("No star is included!");
}

void pp_disk::transform_to_bc(bool verbose)
{
	int n = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();

	tools::transform_to_bc(n, verbose, sim_data);
}

void pp_disk::transform_time(bool verbose)
{
	if (verbose)
	{
		cout << "Transforming the time ... ";
	}

	// Transform the bodies' epochs
	int n = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();
	for (int j = 0; j < n; j++ )
	{
		sim_data->h_epoch[j] *= constants::Gauss;
	}

	if (verbose)
	{
		cout << "done" << endl;
	}
}

void pp_disk::transform_velocity(bool verbose)
{
	if (verbose)
	{
		cout << "Transforming the velocity ... ";
	}

	vec_t* v = sim_data->h_y[1];
	// Transform the bodies' velocities
	int n = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();
	for (int j = 0; j < n; j++ )
	{
		v[j].x /= constants::Gauss;		v[j].y /= constants::Gauss;		v[j].z /= constants::Gauss;
	}

	if (verbose)
	{
		cout << "done" << endl;
	}
}

void pp_disk::create_padding_particle(int k, ttt_t* epoch, body_metadata_t* body_md, param_t* p, vec_t* r, vec_t* v)
{
	body_md[k].id = 0;
	body_names.push_back("Pad_Part");

	body_md[k].body_type = static_cast<body_type_t>(BODY_TYPE_PADDINGPARTICLE);
	epoch[k] = 0.0;
	p[k].mass = 0.0;
	p[k].radius = 0.0;
	p[k].density = 0.0;
	p[k].cd = 0.0;

	body_md[k].mig_type = static_cast<migration_type_t>(MIGRATION_TYPE_NO);
	body_md[k].mig_stop_at = 0.0;

	r[k].x = 1.0e9 + (var_t)rand() / RAND_MAX * 1.0e9;
	r[k].y = r[k].x + (var_t)rand() / RAND_MAX * 1.0e9;
	r[k].z = 0.0;
	r[k].w = 0.0;

	v[k].x = v[k].y = v[k].z = v[k].w = 0.0;
}

void pp_disk::read_body_record(ifstream& input, int k, ttt_t* epoch, body_metadata_t* body_md, param_t* p, vec_t* r, vec_t* v)
{
	int_t	type = 0;
	string	dummy;

	// id
	input >> body_md[k].id;
	// name
	input >> dummy;
	// The names must be less than or equal to 30 chars
	if (dummy.length() > 30)
	{
		dummy = dummy.substr(0, 30);
	}
	body_names.push_back(dummy);
	// body type
	input >> type;
	body_md[k].body_type = static_cast<body_type_t>(type);
	// epoch
	input >> epoch[k];

	// mass, radius density and stokes coefficient
	input >> p[k].mass >> p[k].radius >> p[k].density >> p[k].cd;

	// migration type
	input >> type;
	body_md[k].mig_type = static_cast<migration_type_t>(type);
	// migration stop at
	input >> body_md[k].mig_stop_at;

	// position
	input >> r[k].x >> r[k].y >> r[k].z;
	// velocity
	input >> v[k].x >> v[k].y >> v[k].z;

	r[k].w = v[k].w = 0.0;
}

void pp_disk::load(string& path)
{
	cout << "Loading " << path << " ... ";

	ifstream input(path.c_str());
	if (input) 
	{
		int ns, ngp, nrp, npp, nspl, npl, ntp;
		input >> ns >> ngp >> nrp >> npp >> nspl >> npl >> ntp;
	}
	else 
	{
		throw string("Cannot open " + path + ".");
	}

	vec_t* r = sim_data->h_y[0];
	vec_t* v = sim_data->h_y[1];
	param_t* p = sim_data->h_p;
	body_metadata_t* body_md = sim_data->h_body_md;
	ttt_t* epoch = sim_data->h_epoch;

	int n_SI		= n_bodies->get_n_SI();
	int n_NSI		= n_bodies->get_n_NSI();
	int n_total		= n_bodies->get_n_total();
	int n_prime_SI	= n_bodies->get_n_prime_SI();
	int n_prime_NSI	= n_bodies->get_n_prime_NSI();
	int n_prime_total=n_bodies->get_n_prime_total(); 

    if (input) 
    {
		int i = 0;
		int k = 0;
		for ( ; i < n_SI; i++, k++)
		{
			read_body_record(input, k, epoch, body_md, p, r, v);
		}
        while (use_padded_storage && k < n_prime_SI)
        {
			create_padding_particle(k, epoch, body_md, p, r, v);
            k++;
        }

		for ( ; i < n_SI + n_NSI; i++, k++)
		{
			read_body_record(input, k, epoch, body_md, p, r, v);
		}
		while (use_padded_storage && k < n_prime_SI + n_prime_NSI)
		{
			create_padding_particle(k, epoch, body_md, p, r, v);
			k++;
		}

		for ( ; i < n_total; i++, k++)
		{
			read_body_record(input, k, epoch, body_md, p, r, v);
		}
		while (use_padded_storage && k < n_prime_total)
		{
			create_padding_particle(k, epoch, body_md, p, r, v);
			k++;
		}
        input.close();
	}
	else
    {
		throw string("Cannot open " + path + ".");
	}

	cout << "done" << endl;
}

void pp_disk::print_result_ascii(ostream& sout)
{
	static int int_t_w  =  8;
	static int var_t_w  = 25;

	sout.precision(16);
	sout.setf(ios::right);
	sout.setf(ios::scientific);

	vec_t* r = sim_data->h_y[0];
	vec_t* v = sim_data->h_y[1];
	param_t* p = sim_data->h_p;
	body_metadata_t* body_md = sim_data->h_body_md;

	int n = use_padded_storage ? n_bodies->get_n_prime_total() : n_bodies->get_n_total();
	for (int i = 0; i < n; i++)
    {
		// Skip inactive bodies and padding particles and alike
		if (body_md[i].id <= 0 || body_md[i].body_type >= BODY_TYPE_PADDINGPARTICLE)
		{
			continue;
		}
		sout << setw(int_t_w) << body_md[i].id << SEP                /* id of the body starting from 1                                (int)              */
			 << setw(     30) << body_names[i] << SEP                /* name of the body                                              (string = 30 char) */ 
			 << setw(      2) << body_md[i].body_type << SEP         /* type of the body                                              (int)              */
			 << setw(var_t_w) << t / constants::Gauss << SEP         /* time of the record                           [day]            (double)           */
			 << setw(var_t_w) << p[i].mass << SEP                    /* mass of the body                             [solar mass]     (double)           */
			 << setw(var_t_w) << p[i].radius << SEP                  /* radius of the body                           [AU]             (double)           */
			 << setw(var_t_w) << p[i].density << SEP                 /* density of the body in                       [solar mass/AU3] (double)           */
			 << setw(var_t_w) << p[i].cd << SEP                      /* Stokes drag coefficeint dimensionless                         (double)           */
			 << setw(      2) << body_md[i].mig_type << SEP          /* migration type of the body                                    (int)              */
			 << setw(var_t_w) << body_md[i].mig_stop_at << SEP       /* migration stops at this barycentric distance [AU]             (double)           */
			 << setw(var_t_w) << r[i].x << SEP                       /* body's x-coordiante in barycentric system    [AU]             (double)           */
			 << setw(var_t_w) << r[i].y << SEP                       /* body's y-coordiante in barycentric system    [AU]             (double)           */
			 << setw(var_t_w) << r[i].z << SEP                       /* body's z-coordiante in barycentric system    [AU]             (double)           */
			 << setw(var_t_w) << v[i].x * constants::Gauss << SEP    /* body's x-velocity in baryentric system       [AU/day]         (double)           */
			 << setw(var_t_w) << v[i].y * constants::Gauss << SEP    /* body's y-velocity in barycentric system      [AU/day]         (double)           */
			 << setw(var_t_w) << v[i].z * constants::Gauss << endl;  /* body's z-velocity in barycentric system      [AU/day]         (double)           */
    }
	sout.flush();
}

void pp_disk::print_result_binary(ostream& sout)
{
	throw string("print_result_binary() is not implemented");
}

void pp_disk::print_event_data(ostream& sout, ostream& log_f)
{
	static int int_t_w =  8;
	static int var_t_w = 25;
	static char *e_names[] = {"NONE", "HIT_CENTRUM", "EJECTION", "CLOSE_ENCOUNTER", "COLLISION"};

	sout.precision(16);
	sout.setf(ios::right);
	sout.setf(ios::scientific);

	log_f.precision(16);
	log_f.setf(ios::right);
	log_f.setf(ios::scientific);

	for (unsigned int i = 0; i < sp_events.size(); i++)
	{
		sout << setw(16)      << e_names[sp_events[i].event_name] << SEP
			 << setw(var_t_w) << sp_events[i].t / constants::Gauss << SEP /* time of the event in day */
			 << setw(var_t_w) << sp_events[i].d << SEP
			 << setw(int_t_w) << sp_events[i].id1 << SEP		/* id of the survivor */
			 << setw(int_t_w) << sp_events[i].id2 << SEP		/* id of the merger */
			 << setw(var_t_w) << sp_events[i].p1.mass << SEP	/* parameters of the survivor before the event */
			 << setw(var_t_w) << sp_events[i].p1.density << SEP
			 << setw(var_t_w) << sp_events[i].p1.radius << SEP
			 << setw(var_t_w) << sp_events[i].r1.x << SEP		/* position of the survivor before the event */
			 << setw(var_t_w) << sp_events[i].r1.y << SEP
			 << setw(var_t_w) << sp_events[i].r1.z << SEP
			 << setw(var_t_w) << sp_events[i].v1.x * constants::Gauss << SEP		/* velocity of the survivor before the event */
			 << setw(var_t_w) << sp_events[i].v1.y * constants::Gauss << SEP
			 << setw(var_t_w) << sp_events[i].v1.z * constants::Gauss << SEP
			 << setw(var_t_w) << sp_events[i].p2.mass << SEP	/* parameters of the merger before the event */
			 << setw(var_t_w) << sp_events[i].p2.density << SEP
			 << setw(var_t_w) << sp_events[i].p2.radius << SEP
			 << setw(var_t_w) << sp_events[i].r2.x << SEP		/* position of the merger before the event */
			 << setw(var_t_w) << sp_events[i].r2.y << SEP
			 << setw(var_t_w) << sp_events[i].r2.z << SEP
			 << setw(var_t_w) << sp_events[i].v2.x * constants::Gauss << SEP		/* velocity of the merger before the event */
			 << setw(var_t_w) << sp_events[i].v2.y * constants::Gauss<< SEP
			 << setw(var_t_w) << sp_events[i].v2.z * constants::Gauss<< SEP
			 << setw(var_t_w) << sp_events[i].ps.mass << SEP	/* parameters of the survivor after the event */
			 << setw(var_t_w) << sp_events[i].ps.density << SEP
			 << setw(var_t_w) << sp_events[i].ps.radius << SEP
			 << setw(var_t_w) << sp_events[i].rs.x << SEP		/* position of the survivor after the event */
			 << setw(var_t_w) << sp_events[i].rs.y << SEP
			 << setw(var_t_w) << sp_events[i].rs.z << SEP
			 << setw(var_t_w) << sp_events[i].vs.x * constants::Gauss << SEP		/* velocity of the survivor after the event */
			 << setw(var_t_w) << sp_events[i].vs.y * constants::Gauss << SEP
			 << setw(var_t_w) << sp_events[i].vs.z * constants::Gauss << SEP << endl;
		if (log_f)
		{
			log_f << tools::get_time_stamp() << SEP 
				  << e_names[sp_events[i].event_name] << SEP
			      << setw(int_t_w) << sp_events[i].id1 << SEP
			      << setw(int_t_w) << sp_events[i].id2 << SEP << endl;
		}
	}
	sout.flush();
	log_f.flush();
}

#undef GAS_REDUCTION_THRESHOLD
#undef GAS_INNER_EDGE

#include <stdio.h>

#include "red_type.h"

using namespace std;

void test_n_objects_t()
{
	const char test_set[] = "test_n_objects_t";

	fprintf(stderr, "TEST: %s\n", test_set);

	// Test get_n_total_initial()
	{
		char test_func[] = "get_n_total_initial";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 28;
		int result = n_bodies.get_n_total_initial();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_total_playing()
	{
		char test_func[] = "get_n_total_playing";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 28;
		int result = n_bodies.get_n_total_playing();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		for (int i = 0; i < BODY_TYPE_N; i++)
		{
			n_bodies.inactive[i]++;
		}
		expected = 28;
		result = n_bodies.get_n_total_playing();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		n_bodies.update();
		expected = 21;
		result = n_bodies.get_n_total_playing();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_total_active()
	{
		char test_func[] = "get_n_total_active";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 28;
		int result = n_bodies.get_n_total_active();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		for (int i = 0; i < BODY_TYPE_N; i++)
		{
			n_bodies.inactive[i]++;
		}
		expected = 21;
		result = n_bodies.get_n_total_active();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_total_inactive()
	{
		char test_func[] = "get_n_total_inactive";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 0;
		int result = n_bodies.get_n_total_inactive();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		for (int i = 0; i < BODY_TYPE_N; i++)
		{
			n_bodies.inactive[i]++;
		}
		expected = 7;
		result = n_bodies.get_n_total_inactive();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_total_removed()
	{
		char test_func[] = "get_n_total_removed";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 0;
		int result = n_bodies.get_n_total_removed();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		for (int i = 0; i < BODY_TYPE_N; i++)
		{
			n_bodies.removed[i]++;
		}
		expected = 7;
		result = n_bodies.get_n_total_removed();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test update()
	{
		char test_func[] = "update";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		for (int i = 0; i < BODY_TYPE_N; i++)
		{
			n_bodies.inactive[i]++;
		}
		n_bodies.update();

		int expected = 21;
		int result = n_bodies.get_n_total_playing();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		expected = 21;
		result = n_bodies.get_n_total_active();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		expected = 0;
		result = n_bodies.get_n_total_inactive();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}

		expected = 7;
		result = n_bodies.get_n_total_removed();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_massive()
	{
		char test_func[] = "get_n_massive";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 21;
		int result = n_bodies.get_n_massive();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_SI
	{
		char test_func[] = "get_n_SI";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 1+2+3+4;
		int result = n_bodies.get_n_SI();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_NSI
	{
		char test_func[] = "get_n_NSI";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 5+6;
		int result = n_bodies.get_n_NSI();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_NI
	{
		char test_func[] = "get_n_NI";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 7;
		int result = n_bodies.get_n_NI();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_GD
	{
		char test_func[] = "get_n_GD";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 5+6;
		int result = n_bodies.get_n_GD();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_MT2
	{
		char test_func[] = "get_n_MT2";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 2;
		int result = n_bodies.get_n_MT2();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_n_MT1
	{
		char test_func[] = "get_n_MT1";

		n_objects_t n_bodies(1, 2, 3, 4, 5, 6, 7);
		int expected = 3+4;
		int result = n_bodies.get_n_MT1();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected != result)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n\t\tExpected: %4d,\n\t\t But was: %4d\n", __LINE__, expected, result);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_bound_SI()
	{
		char test_func[] = "get_bound_SI";

		n_objects_t n_bodies(1, 1, 1, 0, 2, 3, 2);

		interaction_bound expected(0, 3, 0, 8);
		interaction_bound result = n_bodies.get_bound_SI();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected.sink.x   != result.sink.x   || expected.sink.y   != result.sink.y ||
			expected.source.x != result.source.x || expected.source.y != result.source.y)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n", __LINE__);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_bound_NSI()
	{
		char test_func[] = "get_bound_NSI";

		n_objects_t n_bodies(1, 1, 1, 0, 2, 3, 2);

		interaction_bound expected(3, 8, 0, 3);
		interaction_bound result = n_bodies.get_bound_NSI();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected.sink.x != result.sink.x     || expected.sink.y != result.sink.y ||
			expected.source.x != result.source.x || expected.source.y != result.source.y)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n", __LINE__);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}

	// Test get_bound_NI()
	{
		char test_func[] = "get_bound_NI";

		n_objects_t n_bodies(1, 1, 1, 0, 2, 3, 2);

		interaction_bound expected(8, 10, 0, 8);
		interaction_bound result = n_bodies.get_bound_NI();

		fprintf(stderr, "%40s(): ", test_func);
		if (expected.sink.x != result.sink.x     || expected.sink.y != result.sink.y ||
			expected.source.x != result.source.x || expected.source.y != result.source.y)
		{
			fprintf(stderr, "FAILED (Line: %4d)\n", __LINE__);
		}
		else
		{
			fprintf(stderr, "PASSED\n");
		}
	}
}

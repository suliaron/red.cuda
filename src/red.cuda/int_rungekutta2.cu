#include "hip/hip_runtime.h"
// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes project
#include "int_rungekutta2.h"
#include "number_of_bodies.h"
#include "nbody_exception.h"
#include "red_macro.h"
#include "red_constants.h"
#include "util.h"

var_t rungekutta2::a[] = {0.0, 1.0/2.0};
var_t rungekutta2::b[] = {0.0, 1.0};
ttt_t rungekutta2::c[] = {0.0, 1.0/2.0};

namespace rk2_kernel
{
// result = a + b_factor * b
static __global__
	void sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		result[tid] = a[tid] + b_factor * b[tid];
		tid += stride;
	}
}
} /* rk2_kernel */

rungekutta2::rungekutta2(pp_disk *ppd, ttt_t dt) :
	integrator(ppd, dt),
	RKOrder(2),
	d_f(2)
{
	name = "Runge-Kutta2";
	short_name = "RK2";

	const int n_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();

	t = ppd->t;
	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_DEVICE_VECTOR((void**) &(d_ytemp[i]), n_total*sizeof(vec_t));
		d_f[i].resize(RKOrder);
		for (int r = 0; r < RKOrder; r++) 
		{
			ALLOCATE_DEVICE_VECTOR((void**) &(d_f[i][r]), n_total * sizeof(vec_t));
		}
	}
}

rungekutta2::~rungekutta2()
{
	for (int i = 0; i < 2; i++)
	{
		for (int r = 0; r < RKOrder; r++) 
		{
			hipFree(d_f[i][r]);
		}
	}
}

void rungekutta2::call_kernel_calc_ytemp_for_fr(int n_var, int r)
{
	for (int i = 0; i < 2; i++)
	{
		var_t *y_n	  = (var_t*)ppd->sim_data->d_y[i];
		var_t *fr	  = (var_t*)d_f[i][r-1];
		var_t* result = (var_t*)d_ytemp[i];

		rk2_kernel::sum_vector<<<grid, block>>>(n_var, y_n, fr, a[r] * dt_try, result);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus)
		{
			throw string("kernel_sum_vector failed");
		}
	}
}

void rungekutta2::call_kernel_calc_y_np1(int n_var)
{
	for (int i = 0; i < 2; i++)
	{
		var_t *y_n	 = (var_t*)ppd->sim_data->d_y[i];
		var_t *y_np1 = (var_t*)ppd->sim_data->d_yout[i];
		var_t *f2	 = (var_t*)d_f[i][1];

		rk2_kernel::sum_vector<<<grid, block>>>(n_var, y_n, f2, b[1] * dt_try, y_np1);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus)
		{
			throw string("kernel_sum_vector failed");
		}
	}
}

ttt_t rungekutta2::step()
{
	// Set the kernel launch parameters
	const int n_body_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();
	const int n_var_total = NDIM * n_body_total;
	calc_grid(n_var_total, THREADS_PER_BLOCK);

	int r = 0;
	ttt_t ttemp = ppd->t + c[r] * dt_try;
	// Calculate initial differentials f1 = f(tn, yn) and store them into d_f[][0]
	for (int i = 0; i < 2; i++)
	{
		ppd->calc_dy(i, r, ttemp, ppd->sim_data->d_y[0], ppd->sim_data->d_y[1], d_f[i][r]);
	}

	r = 1;
	ttemp = ppd->t + c[r] * dt_try;
	call_kernel_calc_ytemp_for_fr(n_var_total, r);

	// Calculate f2 = f(tn + 1/2*h, yn + 1/2*h*f1) = d_f[][1]
	for (int i = 0; i < 2; i++)
	{
		ppd->calc_dy(i, r, ttemp, d_ytemp[0], d_ytemp[1], d_f[i][r]);
	}

	dt_did = dt_try;
	call_kernel_calc_y_np1(n_var_total);

	update_counters(1);

	ppd->t += dt_did;
	for (int i = 0; i < 2; i++)
	{
		swap(ppd->sim_data->d_yout[i], ppd->sim_data->d_y[i]);
	}

	return dt_did;
}

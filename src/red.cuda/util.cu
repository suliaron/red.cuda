#include "util.h"
#include "nbody_exception.h"
#include "red_type.h"
#include "red_macro.h"

void allocate_device_vector(void **d_ptr, size_t size, const char *file, int line)
{
	hipError_t cudaStatus = hipSuccess;
	hipMalloc(d_ptr, size);
	cudaStatus = HandleError(cudaStatus, file, line);
	if (hipSuccess != cudaStatus)
	{
		throw nbody_exception("hipMalloc failed", cudaStatus);
	}
}

void copy_vector_to_device(void* dst, const void *src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw nbody_exception("hipMemcpy failed (copy_vector_to_device)", cudaStatus);
	}
}

void copy_vector_to_host(void* dst, const void *src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw nbody_exception("hipMemcpy failed (copy_vector_to_host)", cudaStatus);
	}
}

void copy_constant_to_device(const void* dst, const void *src, size_t count)
{
	hipMemcpyToSymbol(HIP_SYMBOL(dst), src, count);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw nbody_exception("hipMemcpyToSymbol failed (copy_constant_to_device)", cudaStatus);
	}
}

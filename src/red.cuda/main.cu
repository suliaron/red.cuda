#include "hip/hip_runtime.h"
// includes system
#include <cmath>
#include <ctime>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <memory>

// includes CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

// includes Thrust
#ifdef __GNUC__
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#else
#include "thrust\device_ptr.h"
#include "thrust\fill.h"
#include "thrust\extrema.h"
#endif

// includes project
#include "int_euler.h"
#include "int_rungekutta2.h"
#include "int_rungekutta4.h"
#include "parameter.h"
#include "redutilcu.h"
#include "nbody_exception.h"
#include "options.h"
#include "red_type.h"
#include "red_constants.h"

using namespace std;
using namespace redutilcu;

void open_streams(const options& opt, const integrator* intgr, ostream** result_f, ostream** info_f, ostream** event_f, ostream** log_f)
{
	string path;

	{
		string ext = file::get_extension(opt.result_filename);

		string adapt = (opt.param->adaptive == true ? "_a_" : "_");
		string result_filename = file::get_filename_without_ext(opt.result_filename) + adapt + intgr->short_name + "." + ext;
		path = file::combine_path(opt.printout_dir, result_filename);
		*result_f = new ofstream(path.c_str(), ios::out);
	}

	path = file::combine_path(opt.printout_dir, opt.info_filename);
	*info_f = new ofstream(path.c_str(), ios::out);

	path = file::combine_path(opt.printout_dir, opt.event_filename);
	*event_f = new ofstream(path.c_str(), ios::out);

	path = file::combine_path(opt.printout_dir, opt.log_filename);
	*log_f = new ofstream(path.c_str(), ios::out);
}

void print_info(ostream& sout, const pp_disk* ppd, integrator *intgr, ttt_t dt, clock_t* sum_time_of_steps, clock_t* time_of_one_step, time_t* time_info_start)
{
	cout.setf(ios::right);
	cout.setf(ios::scientific);

	sout.setf(ios::right);
	sout.setf(ios::scientific);

	number_of_bodies* nb = ppd->n_bodies; 

	*time_info_start = clock();
	cout << tools::get_time_stamp() << " t: " << setprecision(6) << setw(12) << ppd->t 
		 << ", dt: " << setprecision(6) << setw(12)  << dt;
	cout << ", dT_cpu: " << setprecision(3) << setw(10) << *time_of_one_step / (double)CLOCKS_PER_SEC << " s";
	cout << ", dT_avg: " << setprecision(3) << setw(10) << (*sum_time_of_steps / (double)CLOCKS_PER_SEC) / intgr->get_n_passed_step() << " s";
	cout << ", Nc: " << setw(5) << ppd->n_collision[  EVENT_COUNTER_NAME_TOTAL]
	     << ", Ne: " << setw(5) << ppd->n_ejection[   EVENT_COUNTER_NAME_TOTAL]
		 << ", Nh: " << setw(5) << ppd->n_hit_centrum[EVENT_COUNTER_NAME_TOTAL]
		 << ", N : " << setw(6) << nb->get_n_total() << "(" << setw(6) << nb->get_n_total_inactive() << ")" << endl;

	sout << tools::get_time_stamp() << " t: " << setprecision(6) << setw(12) << ppd->t 
		 << ", dt: " << setprecision(6) << setw(12)  << dt;
	sout << ", dT_cpu: " << setprecision(3) << setw(10) << *time_of_one_step / (double)CLOCKS_PER_SEC << " s";
	sout << ", dT_avg: " << setprecision(3) << setw(10) << (*sum_time_of_steps / (double)CLOCKS_PER_SEC) / intgr->get_n_passed_step() << " s";
	sout << ", Nc: " << setw(5) << ppd->n_collision[  EVENT_COUNTER_NAME_TOTAL]
	     << ", Ne: " << setw(5) << ppd->n_ejection[   EVENT_COUNTER_NAME_TOTAL]
		 << ", Nh: " << setw(5) << ppd->n_hit_centrum[EVENT_COUNTER_NAME_TOTAL]
		 << ", N : " << setw(6) << nb->get_n_total() << "(" << setw(6) << nb->get_n_total_inactive() << ")"
	     << ", N_st: " << setw(5) << nb->n_s   << "(" << setw(5) << nb->n_i_s << ")"
		 << ", N_gp: " << setw(5) << nb->n_gp  << "(" << setw(5) << nb->n_i_gp << ")"
		 << ", N_rp: " << setw(5) << nb->n_rp  << "(" << setw(5) << nb->n_i_rp << ")"
		 << ", N_pp: " << setw(5) << nb->n_pp  << "(" << setw(5) << nb->n_i_pp << ")"
		 << ", N_sp: " << setw(5) << nb->n_spl << "(" << setw(5) << nb->n_i_spl << ")"
		 << ", N_pl: " << setw(5) << nb->n_pl  << "(" << setw(5) << nb->n_i_pl << ")"
		 << ", N_tp: " << setw(5) << nb->n_tp  << "(" << setw(5) << nb->n_i_tp << ")" << endl;	
}

ttt_t step(integrator *intgr, clock_t* sum_time_of_steps, clock_t* time_of_one_step)
{
	clock_t start_of_step = clock();
	ttt_t dt = intgr->step();
	clock_t end_of_step = clock();

	*time_of_one_step = (end_of_step - start_of_step);
	*sum_time_of_steps += *time_of_one_step;

	return dt;
}

//http://stackoverflow.com/questions/11666049/cuda-kernel-results-different-in-release-mode
//http://developer.download.nvidia.com/assets/cuda/files/NVIDIA-CUDA-Floating-Point.pdf

int main(int argc, const char** argv, const char** env)
{
	time_t start = time(NULL);

	ostream* result_f = 0x0;
	ostream* info_f = 0x0;
	ostream* event_f = 0x0;
	ostream* log_f = 0x0;
	try
	{
		options opt = options(argc, argv);
		pp_disk *ppd = opt.create_pp_disk();
		integrator *intgr = opt.create_integrator(ppd, 0.001);
		open_streams(opt, intgr, &result_f, &info_f, &event_f, &log_f);

		file::log_start_cmd(*log_f, argc, argv, env);
		if (opt.verbose)
		{
			file::log_start_cmd(cout, argc, argv, env);
			if (!opt.cpu)
			{
				device_query(cout, opt.id_a_dev);
			}
		}
		if (!opt.cpu)
		{
			device_query(*log_f, opt.id_a_dev);
		}

		ttt_t ps = 0;
		ttt_t dt = 0;
		clock_t sum_time_of_steps = 0.0;
		clock_t time_of_one_step  = 0.0;

		time_t time_info_start = clock();

		ppd->print_result_ascii(*result_f);
		while (ppd->t <= opt.param->stop_time)
		{
			if (fabs(ps) >= opt.param->output_interval)
			{
				ps = 0.0;
				if (!opt.cpu)
				{
					ppd->copy_to_host();
				}
				ppd->print_result_ascii(*result_f);
			}

			if (ppd->check_for_ejection_hit_centrum())
			{
				ppd->print_event_data(*event_f, *log_f);
				ppd->clear_event_counter();
			}

			dt = step(intgr, &sum_time_of_steps, &time_of_one_step);
			ps += fabs(dt);

			if (opt.param->thrshld[THRESHOLD_COLLISION_FACTOR] > 0.0 && ppd->check_for_collision())
			{
				ppd->print_event_data(*event_f, *log_f);
				ppd->clear_event_counter();
			}

			if (ppd->check_for_rebuild_vectors(8))
			{
				file::log_rebuild_vectors(*log_f, ppd->t);
			}

			if ((clock() - time_info_start) / (double)CLOCKS_PER_SEC > 5.0) 
			{
				print_info(*info_f, ppd, intgr, dt, &sum_time_of_steps, &time_of_one_step, &time_info_start);
			}
		} /* while */
		print_info(*info_f, ppd, intgr, dt, &sum_time_of_steps, &time_of_one_step, &time_info_start);

		// To avoid duplicate save at the end of the simulation
		if (ps > 0.0)
		{
			if (!opt.cpu) ppd->copy_to_host();
			ppd->print_result_ascii(*result_f);
		}

	} /* try */
	catch (const nbody_exception& ex)
	{
		if (0x0 != log_f)
		{
			file::log_message(*log_f, "Error: " + string(ex.what()));
		}
		cerr << "Error: " << ex.what() << endl;
	}
	catch (const string& msg)
	{
		if (0x0 != log_f)
		{
			file::log_message(*log_f, "Error: " + msg);
		}
		cerr << "Error: " << msg << endl;
	}
	if (0x0 != log_f)
	{
		file::log_message(*log_f, " Total time: " + tools::convert_time_t(time(NULL) - start) + " s");
	}
	cout << "Total time: " << time(NULL) - start << " s" << endl;

	// Needed by nvprof.exe
	hipDeviceReset();

	return (EXIT_SUCCESS);
}

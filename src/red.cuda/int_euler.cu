#include "hip/hip_runtime.h"
// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes project
#include "int_euler.h"
#include "number_of_bodies.h"
#include "nbody_exception.h"
#include "red_macro.h"
#include "red_constants.h"
#include "util.h"

namespace euler_kernel
{
// result = a + b_factor * b
static __global__
	void sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		result[tid] = a[tid] + b_factor * b[tid];
		tid += stride;
	}
}
} /* euler_kernel */

void euler::cpu_calc_y_np1(int n_var)
{
	for (int i = 0; i < 2; i++)
	{	
		var_t *y_n	 = (var_t*)ppd->sim_data->y[i];
		var_t *y_np1 = (var_t*)ppd->sim_data->yout[i];
		var_t *f0	 = (var_t*)dydx[i];

		for (int j = 0; j < n_var; j++)
		{
			y_np1[j] = y_n[j] + dt_try * f0[j];
		}
	}
}

euler::euler(pp_disk *ppd, ttt_t dt, bool cpu) :
	integrator(ppd, dt, cpu),
	dydx(2)
{
	name = "Euler";
	short_name = "E";

	const int n_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();

	t = ppd->t;
	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_VECTOR((void**)&(dydx[i]), n_total*sizeof(vec_t), cpu);
	}
}

euler::~euler()
{
	if (!cpu)
	{
		hipFree(dydx[0]);
		hipFree(dydx[1]);
	}
	else
	{
		delete[] dydx[0];
		delete[] dydx[1];
	}
}

void euler::call_kernel_calc_y_np1(int n_var)
{
	for (int i = 0; i < 2; i++)
	{	
		var_t *y_n	 = (var_t*)ppd->sim_data->d_y[i];
		var_t *y_np1 = (var_t*)ppd->sim_data->d_yout[i];
		var_t *f0	 = (var_t*)dydx[i];

		euler_kernel::sum_vector<<<grid, block>>>(n_var, y_n, f0, dt_try, y_np1);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus) 
		{
			throw nbody_exception("kernel_sum_vector failed", cudaStatus);
		}
	}
}

void euler::calc_y_np1(int n_var_total)
{
	if (!cpu)
	{
		call_kernel_calc_y_np1(n_var_total);
	}
	else
	{
		cpu_calc_y_np1(n_var_total);
	}
}

ttt_t euler::step()
{
	// Set the kernel launch parameters
	const int n_body_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();
	const int n_var_total = NDIM * n_body_total;
	if (!cpu)
	{
		calc_grid(n_var_total, THREADS_PER_BLOCK);
	}

	t = ppd->t;
	// Calculate initial differentials and store them into f
	for (int i = 0; i < 2; i++)
	{
		const vec_t *r = cpu ? ppd->sim_data->y[0] : ppd->sim_data->d_y[0];
		const vec_t *v = cpu ? ppd->sim_data->y[1] : ppd->sim_data->d_y[1];

		ppd->calc_dydx(i, 0, t, r, v, dydx[i]);
	}
	calc_y_np1(n_var_total);

	dt_did = dt_try;
	dt_next = dt_try;

	update_counters(1);

	ppd->t += dt_did;
	ppd->swap();

	return dt_did;
}

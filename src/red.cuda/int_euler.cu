#include "hip/hip_runtime.h"
// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes project
#include "int_euler.h"
#include "number_of_bodies.h"
#include "nbody_exception.h"
#include "red_macro.h"
#include "red_constants.h"
#include "util.h"

namespace euler_kernel
{
// result = a + b_factor * b
static __global__
	void sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		result[tid] = a[tid] + b_factor * b[tid];
		tid += stride;
	}
}
} /* euler_kernel */

void euler::cpu_sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	for (int tid = 0; tid < n; tid++)
	{
		result[tid] = a[tid] + b_factor * b[tid];
	}
}

euler::euler(pp_disk *ppd, ttt_t dt, bool cpu) :
	integrator(ppd, dt, cpu),
	dydx(2)
{
	name = "Euler";
	short_name = "E";

	const int n_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();

	t = ppd->t;
	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_VECTOR((void**)&(dydx[i]), n_total*sizeof(vec_t), cpu);
	}
}

euler::~euler()
{
	FREE_VECTOR(dydx[0], cpu);
	FREE_VECTOR(dydx[1], cpu);
}

void euler::calc_y_np1(int n_var)
{
	for (int i = 0; i < 2; i++)
	{	
		var_t *y_n	 = (var_t*)ppd->sim_data->y[i];
		var_t *y_np1 = (var_t*)ppd->sim_data->yout[i];
		var_t *f0	 = (var_t*)dydx[i];

		if (!cpu)
		{
			euler_kernel::sum_vector<<<grid, block>>>(n_var, y_n, f0, dt_try, y_np1);

			hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
			if (hipSuccess != cudaStatus) 
			{
				throw nbody_exception("euler_kernel::sum_vector failed", cudaStatus);
			}
		}
		else
		{
			cpu_sum_vector(n_var, y_n, f0, dt_try, y_np1);
		}
	}
}

ttt_t euler::step()
{
	// Set the kernel launch parameters
	const int n_body_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();
	const int n_var_total = NDIM * n_body_total;
	if (!cpu)
	{
		calc_grid(n_var_total, THREADS_PER_BLOCK);
	}

	t = ppd->t;
	// Calculate initial differentials and store them into dydx
	const vec_t *coor = ppd->sim_data->y[0];
	const vec_t *velo = ppd->sim_data->y[1];
	for (int i = 0; i < 2; i++)
	{
		ppd->calc_dydx(i, 0, t, coor, velo, dydx[i]);
	}
	calc_y_np1(n_var_total);

	dt_did = dt_try;
	dt_next = dt_try;

	update_counters(1);

	ppd->t += dt_did;
	ppd->swap();

	return dt_did;
}

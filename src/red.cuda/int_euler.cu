#include "hip/hip_runtime.h"
// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// DEBUG --
#include <sstream>		// ostringstream
#include "redutilcu.h"
using namespace redutilcu;
// DEBUG --

// includes project
#include "int_euler.h"
#include "number_of_bodies.h"
#include "nbody_exception.h"
#include "red_macro.h"
#include "red_constants.h"
#include "util.h"

namespace euler_kernel
{
// result = a + b_factor * b
static __global__
	void sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		result[tid] = a[tid] + b_factor * b[tid];
		tid += stride;
	}
}
} /* namespace euler_kernel */

void euler::cpu_sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	for (int tid = 0; tid < n; tid++)
	{
		result[tid] = a[tid] + b_factor * b[tid];
	}
}

euler::euler(pp_disk *ppd, ttt_t dt, computing_device_t comp_dev) :
	integrator(ppd, dt, false, 0.0, 1, comp_dev)
{
	name = "Euler";
	short_name = "E";

	order = 1;
}

euler::~euler()
{
}

void euler::calc_y_np1(int n_var)
{
	for (int i = 0; i < 2; i++)
	{	
		var_t *y_n	 = (var_t*)ppd->sim_data->y[i];
		var_t *y_np1 = (var_t*)ppd->sim_data->yout[i];
		var_t *f0	 = (var_t*)dydx[i][0];

		if (COMPUTING_DEVICE_GPU == comp_dev)
		{
			euler_kernel::sum_vector<<<grid, block>>>(n_var, y_n, f0, dt_try, y_np1);

			hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
			if (hipSuccess != cudaStatus) 
			{
				throw nbody_exception("euler_kernel::sum_vector failed", cudaStatus);
			}
		}
		else
		{
			cpu_sum_vector(n_var, y_n, f0, dt_try, y_np1);
		}
// DEBUG CODE BEGIN
//ostringstream convert;	// stream used for the conversion
//convert << i;
//string i_str = convert.str();
//string dev = (comp_dev == COMPUTING_DEVICE_CPU ? "cpu" : "gpu");
//string filename = "y_np1_" + i_str + "_" + dev + ".txt";
//string path = file::combine_path("C:\\Work\\Projects\\red.cuda\\TestRun\\InputTest\\TwoBody_GPU_change2_CPU", filename);
//print_array(path, n_var, y_np1, comp_dev);
// DEBUG CODE END
	}
}

ttt_t euler::step()
{
	const int n_body_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();
	const int n_var_total = NDIM * n_body_total;

	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		// Set the kernel launch parameters
		calc_grid(n_var_total, THREADS_PER_BLOCK);
	}

	t = ppd->t;
	// Calculate initial differentials and store them into dydx
	const vec_t *coor = ppd->sim_data->y[0];
	const vec_t *velo = ppd->sim_data->y[1];
// DEBUG CODE BEGIN
//string dev = (comp_dev == COMPUTING_DEVICE_CPU ? "cpu" : "gpu");
//string filename = "coor_" + dev + ".txt";
//string path = file::combine_path("C:\\Work\\Projects\\red.cuda\\TestRun\\InputTest\\TwoBody_GPU_change2_CPU", filename);
//print_array(path, n_var_total, (var_t*)(coor), comp_dev);
//filename = "velo_" + dev + ".txt";
//path = file::combine_path("C:\\Work\\Projects\\red.cuda\\TestRun\\InputTest\\TwoBody_GPU_change2_CPU", filename);
//print_array(path, n_var_total, (var_t*)(velo), comp_dev);
// DEBUG CODE END

	for (int i = 0; i < 2; i++)
	{
		ppd->calc_dydx(i, 0, t, coor, velo, dydx[i][0]);
// DEBUG CODE BEGIN
//ostringstream convert;	// stream used for the conversion
//convert << i;
//string i_str = convert.str();
//string dev = (comp_dev == COMPUTING_DEVICE_CPU ? "cpu" : "gpu");
//string filename = "dydx_" + i_str + "_" + dev + ".txt";
//string path = file::combine_path("C:\\Work\\Projects\\red.cuda\\TestRun\\InputTest\\TwoBody_GPU_change2_CPU", filename);
//print_array(path, n_var_total, (var_t*)(dydx[i][0]), comp_dev);
// DEBUG CODE END
	}

	calc_y_np1(n_var_total);

	dt_did = dt_try;
	dt_next = dt_try;

	update_counters(1);

	ppd->t += dt_did;
	ppd->swap();

	return dt_did;
}

#include "hip/hip_runtime.h"
// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes project
#include "int_euler.h"
#include "number_of_bodies.h"
#include "nbody_exception.h"
#include "red_macro.h"
#include "red_constants.h"
#include "util.h"

namespace euler_kernel
{
// result = a + b_factor * b
static __global__
	void sum_vector(int n, const var_t* a, const var_t* b, var_t b_factor, var_t* result)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		result[tid] = a[tid] + b_factor * b[tid];
		tid += stride;
	}
}
} /* euler_kernel */

euler::euler(pp_disk *ppd, ttt_t dt) :
	integrator(ppd, dt),
	d_df(2)
{
	name = "Euler";
	short_name = "E";

	const int n_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();

	t = ppd->t;
	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_DEVICE_VECTOR((void**)&(d_df[i]), n_total*sizeof(vec_t));
	}
}

euler::~euler()
{
	hipFree(d_df[0]);
	hipFree(d_df[1]);
}

void euler::call_kernel_calc_y_np1(int n_var)
{
	for (int i = 0; i < 2; i++)
	{	
		var_t *y_n	 = (var_t*)ppd->sim_data->d_y[i];
		var_t *y_np1 = (var_t*)ppd->sim_data->d_yout[i];
		var_t *f0	 = (var_t*)d_df[i];

		euler_kernel::sum_vector<<<grid, block>>>(n_var, y_n, f0, dt_try, y_np1);
		hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
		if (hipSuccess != cudaStatus) 
		{
			throw nbody_exception("kernel_sum_vector failed", cudaStatus);
		}
	}
}

ttt_t euler::step()
{
	// Set the kernel launch parameters
	const int n_body_total = ppd->get_ups() ? ppd->n_bodies->get_n_prime_total() : ppd->n_bodies->get_n_total();
	const int n_var_total = NDIM * n_body_total;
	calc_grid(n_var_total, THREADS_PER_BLOCK);

	t = ppd->t;
	// Calculate initial differentials and store them into d_dy
	for (int i = 0; i < 2; i++)
	{
		ppd->calc_dy(i, 0, t, ppd->sim_data->d_y[0], ppd->sim_data->d_y[1], d_df[i]);
	}
	call_kernel_calc_y_np1(n_var_total);

	dt_did = dt_try;
	dt_next = dt_try;

	update_counters(1);

	ppd->t += dt_did;
	for (int i = 0; i < 2; i++)
	{
		swap(ppd->sim_data->d_yout[i], ppd->sim_data->d_y[i]);
	}

	return dt_did;
}

#include "hip/hip_runtime.h"
// includes system
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <iostream>
#include <stdlib.h>
#include <string>

// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes Thrust
#include "thrust\device_ptr.h"
#include "thrust\fill.h"
#include "thrust\extrema.h"

// includes project
#include "number_of_bodies.h"
#include "red_type.h"
#include "red_macro.h"
#include "util.h"


using namespace std;

static __global__
	void	kernel_calc_grav_accel
	(
		ttt_t t, 
		interaction_bound int_bound, 
		const body_metadata_t* body_md, 
		const param_t* p, 
		const vec_t* r, 
		const vec_t* v, 
		vec_t* a,
		event_data_t* events,
		int *event_counter
	)
{
	const int i = int_bound.sink.x + blockIdx.x * blockDim.x + threadIdx.x;

	if (i < int_bound.sink.y)
	{
		a[i].x = 0.0;
		a[i].y = 0.0;
		a[i].z = 0.0;
		a[i].w = 0.0;
		if (body_md[i].id > 0)
		{
			vec_t dVec = {0.0, 0.0, 0.0, 0.0};
			for (int j = int_bound.source.x; j < int_bound.source.y; j++) 
			{
				/* Skip the body with the same index and those which are inactive ie. id < 0 */
				if (i == j || body_md[j].id < 0)
				{
					continue;
				}
				// 3 FLOP
				dVec.x = r[j].x - r[i].x;
				dVec.y = r[j].y - r[i].y;
				dVec.z = r[j].z - r[i].z;
				// 5 FLOP
				dVec.w = SQR(dVec.x) + SQR(dVec.y) + SQR(dVec.z);	// = r2
				// 20 FLOP
				var_t d = sqrt(dVec.w);								// = r
				// 2 FLOP
				dVec.w = p[j].mass / (d*dVec.w);
				// 6 FLOP
				a[i].x += dVec.w * dVec.x;
				a[i].y += dVec.w * dVec.y;
				a[i].z += dVec.w * dVec.z;

				// Check for collision - ignore the star (i > 0 criterium)
				// The data of the collision will be stored for the body with the greater index (test particles can collide with massive bodies)
				// If i < j is the condition than test particles can not collide with massive bodies
				if (i > 0 && i > j && d < /* dc_threshold[THRESHOLD_COLLISION_FACTOR] */ 5.0 * (p[i].radius + p[j].radius))
				{
					unsigned int k = atomicAdd(event_counter, 1);

					int survivIdx = i;
					int mergerIdx = j;
					if (p[mergerIdx].mass > p[survivIdx].mass)
					{
						int t = survivIdx;
						survivIdx = mergerIdx;
						mergerIdx = t;
					}
					//printf("t = %20.10le d = %20.10le %d. COLLISION detected: id: %5d id: %5d\n", t, d, k+1, body_md[survivIdx].id, body_md[mergerIdx].id);

					events[k].event_name = EVENT_NAME_COLLISION;
					events[k].d = d;
					events[k].t = t;
					events[k].id.x = body_md[survivIdx].id;
					events[k].id.y = body_md[mergerIdx].id;
					events[k].idx.x = survivIdx;
					events[k].idx.y = mergerIdx;
					events[k].r1 = r[survivIdx];
					events[k].v1 = v[survivIdx];
					events[k].r2 = r[mergerIdx];
					events[k].v2 = v[mergerIdx];
				}
			}
			// 36 FLOP
			// With the used time unit k = 1
			//a[i].x *= K2;
			//a[i].y *= K2;
			//a[i].z *= K2;
		}
	}
}

__global__
	void kernel_print_array(int n, const var_t* v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		printf("v[%4d] : %20.16lf\n", i, v[i]);
	}
}

__global__
	void set_element_of_array(int n, int idx, var_t* v, var_t value)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n && idx == i)
	{
		v[idx] = value;
	}
}

__global__
	void kernel_print_vector(int n, const vec_t* v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		printf("v[%4d].x : %20.16lf\n", i, v[i].x);
		printf("v[%4d].y : %20.16lf\n", i, v[i].y);
		printf("v[%4d].z : %20.16lf\n", i, v[i].z);
		printf("v[%4d].w : %20.16lf\n", i, v[i].w);
	}
}

__global__
	void kernel_print_position(int n, const vec_t* r)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n)
	{
		//printf("r[%4d]: %f\n", tid, r[tid]);
		printf("r[%4d].x: %f\n", tid, r[tid].x);
		printf("r[%4d].y: %f\n", tid, r[tid].y);
		printf("r[%4d].z: %f\n", tid, r[tid].z);
		printf("r[%4d].w: %f\n", tid, r[tid].w);
	}
}

void test_print_position(int n, const vec_t* r)
{
	for (int tid = 0; tid < n; tid++)
	{
		//printf("r[%4d]: %f\n", tid, r[tid]);
		printf("r[%4d].x: %f\n", tid, r[tid].x);
		printf("r[%4d].y: %f\n", tid, r[tid].y);
		printf("r[%4d].z: %f\n", tid, r[tid].z);
		printf("r[%4d].w: %f\n", tid, r[tid].w);
	}
}

void allocate_device_vector(void **d_ptr, size_t size)
{
	hipMalloc(d_ptr, size);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw string("hipMalloc failed");
	}
}


#if 0
// Study and test how a struct is placed into the memory
// Study and test how an array of struct is placed into the memory
int main(int argc, const char** argv)
{
	sim_data_t sim_data;

	sim_data.y.resize(2);
	sim_data.y[0] = new vec_t[8];
	sim_data.y[1] = new vec_t[8];

	sim_data.d_y.resize(2);

	var_t xmax =  1.0;
	var_t xmin = -1.0;
	for (int i = 0; i < 8; i++)
	{
		sim_data.y[0][i].x = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		sim_data.y[0][i].y = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		sim_data.y[0][i].z = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		sim_data.y[0][i].w = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
	}
	test_print_position(8, sim_data.y[0]);

	// Allocate device pointer.
	hipMalloc((void**) &(sim_data.d_y[0]), 8*sizeof(vec_t));
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMalloc failed" << endl;
		return EXIT_FAILURE;
	}
	// Copy pointer content (position and mass) from host to device.
	hipMemcpy(sim_data.d_y[0], sim_data.y[0], 8*sizeof(vec_t), hipMemcpyHostToDevice);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMemcpy failed" << endl;
		return EXIT_FAILURE;
	}
	
	kernel_print_position<<<1, 8>>>(8, sim_data.d_y[0]);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "kernel_print_position failed" << endl;
		return EXIT_FAILURE;
	}
	hipDeviceSynchronize();

	// Allocate pointer.
	vec_t*	v = 0;
	v = (vec_t*)malloc(8 * sizeof(vec_t));
	memset(v, 0, 8 * sizeof(vec_t));

	// Allocate device pointer.
	vec_t*	d_v = 0;
	hipMalloc((void**) &(d_v), 8*sizeof(vec_t));
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMalloc failed" << endl;
		return EXIT_FAILURE;
	}

	// Copy pointer content from host to device.
	hipMemcpy(d_v, v, 8*sizeof(vec_t), hipMemcpyHostToDevice);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMemcpy failed" << endl;
		return EXIT_FAILURE;
	}

	kernel_print_vector<<<1, 8>>>(8, d_v);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "kernel_print_vector failed" << endl;
		return EXIT_FAILURE;
	}
	hipDeviceSynchronize();

	free(v);
	hipFree(d_v);

	delete[] sim_data.y[0];
	delete[] sim_data.y[1];
	hipFree(sim_data.d_y[0]);

	return EXIT_SUCCESS;
}
#endif

#if 0
// Study and test the vector<vector <vec_t*> > type
int main(int argc, const char** argv)
{
	vector<vector <vec_t*> >	d_f(8);		// size of the outer vector container

	for (int i = 0; i < 8; i++)
	{
		d_f[i].resize(2);					// size of the inner vector container
		for (int j = 0; j < 2; j++)
		{
			d_f[i][j] = new vec_t[4];		// allocate 4 vec_t type element for each i, j pair
		}
	}
}
#endif

#if 0
// Study how to wrap a vec_t* into thrust vector to find the maximal element
// howto find the index of the maximum element
int main(int argc, const char** argv)
{
	{
		int data[6] = {1, 0, 2, 2, 1, 3};
		int *result = thrust::max_element(data, data + 6);
		printf("result: %p\n", result);
		printf("*result: %d\n", *result);
		int i = 0;
	}

	//! Holds the leading local truncation error for each variable
	vector<var_t*> d_err(1);

	// Create a raw pointer to device memory
	allocate_device_vector((void**)&d_err[0], 8 * sizeof(var_t));

	set_element_of_array<<<1, 8>>>(8, 2, d_err[0], 3.1415926535897932384626433832795);
	set_element_of_array<<<1, 8>>>(8, 5, d_err[0], 5.987654321);
	kernel_print_array<<<1, 8>>>(8, d_err[0]);
	hipDeviceSynchronize();

	// Wrap raw pointer with a device_ptr
	thrust::device_ptr<var_t> d_ptr(d_err[0]);

	printf("d_ptr: %p\n", d_ptr.get());

	thrust::device_ptr<var_t> d_ptr_max_element = thrust::max_element(d_ptr, d_ptr + 8);

	var_t max_element = 0.0;
	// Copy the max element from device memory to host memory
	hipMemcpy((void*)&max_element, (void*)d_ptr_max_element.get(), sizeof(var_t), hipMemcpyDeviceToHost);
	printf("Value of max_element: %lf\n", max_element);
	
	printf("d_ptr_max_element: %p\n", d_ptr_max_element.get());
	int idx_of_max_element = (d_ptr_max_element.get() - d_ptr.get());
	cout << "idx_of_max_element: " << idx_of_max_element << endl;

	max_element = *thrust::max_element(d_ptr, d_ptr + 8);
	cout << "max_element: " << max_element << endl;

	// Use device_ptr in thrust algorithms
	thrust::fill(d_ptr, d_ptr + 8, (var_t)0.0);

	kernel_print_array<<<1, 8>>>(8, d_err[0]);
	hipDeviceSynchronize();

	hipFree(d_err[0]);
}
#endif

// Measure the execution time of the kernel computing the gravitational acceleleration

ttt_t t;
//interaction_bound int_bound;
body_metadata_t* body_md;
param_t* p;
vec_t* r;
vec_t* v;
vec_t* a;
event_data_t* events;
event_data_t* d_events;
int *event_counter;
int *d_event_counter;

void allocate_storage(const number_of_bodies *n_bodies, sim_data_t *sim_data)
{
	const int nBody = n_bodies->total;

	sim_data = new sim_data_t;

	sim_data->y.resize(2);
	for (int i = 0; i < 2; i++)
	{
		sim_data->y[i]	= new vec_t[nBody];
	}
	sim_data->p	= new param_t[nBody];
	sim_data->body_md	= new body_metadata_t[nBody];
	sim_data->epoch		= new ttt_t[nBody];

	events = new event_data_t[nBody];

	sim_data->d_y.resize(2);
	sim_data->d_yout.resize(2);
	// Allocate device pointer.
	for (int i = 0; i < 2; i++)
	{
		ALLOCATE_DEVICE_VECTOR((void **)&(sim_data->d_y[i]),	nBody*sizeof(vec_t));
		ALLOCATE_DEVICE_VECTOR((void **)&(sim_data->d_yout[i]),	nBody*sizeof(vec_t));
	}
	ALLOCATE_DEVICE_VECTOR((void **)&(sim_data->d_p),			nBody*sizeof(param_t));
	ALLOCATE_DEVICE_VECTOR((void **)&(sim_data->d_body_md),		nBody*sizeof(body_metadata_t));
	ALLOCATE_DEVICE_VECTOR((void **)&(sim_data->d_epoch),		nBody*sizeof(ttt_t));

	ALLOCATE_DEVICE_VECTOR((void **)&d_events,					nBody*sizeof(event_data_t));
	ALLOCATE_DEVICE_VECTOR((void **)&d_event_counter,				1*sizeof(int));
}

void copy_to_device(const number_of_bodies *n_bodies, const sim_data_t *sim_data)
{
	const int n = n_bodies->total;

	for (int i = 0; i < 2; i++)
	{
		copy_vector_to_device((void *)sim_data->d_y[i],	(void *)sim_data->y[i],		n*sizeof(vec_t));
	}
	copy_vector_to_device((void *)sim_data->d_p,		(void *)sim_data->p,		n*sizeof(param_t));
	copy_vector_to_device((void *)sim_data->d_body_md,	(void *)sim_data->body_md,	n*sizeof(body_metadata_t));
	copy_vector_to_device((void *)sim_data->d_epoch,	(void *)sim_data->epoch,	n*sizeof(ttt_t));
	copy_vector_to_device((void *)d_event_counter,		(void *)&event_counter,		1*sizeof(int));
}

void populate_data(const number_of_bodies *n_bodies, sim_data_t *sim_data)
{
}

void deallocate_storage(sim_data_t *sim_data)
{
	for (int i = 0; i < 2; i++)
	{
		delete[] sim_data->y[i];
	}
	delete[] sim_data->p;
	delete[] sim_data->body_md;
	delete[] sim_data->epoch;
	delete[] events;

	for (int i = 0; i < 2; i++)
	{
		hipFree(sim_data->d_y[i]);
		hipFree(sim_data->d_yout[i]);
	}
	hipFree(sim_data->d_p);
	hipFree(sim_data->d_body_md);
	hipFree(sim_data->d_epoch);
	hipFree(d_events);
	hipFree(d_event_counter);

	delete sim_data;
}

int main(int argc, const char** argv)
{
	number_of_bodies n_bodies = number_of_bodies(1, 0, 0, 5000, 0, 0, 5000);

	sim_data_t *sim_data = 0x0;
	t = 0.0;
	*event_counter = 0;

	allocate_storage(&n_bodies, sim_data);
	populate_data(&n_bodies, sim_data);
	copy_to_device(&n_bodies, sim_data);



	deallocate_storage(sim_data);
}

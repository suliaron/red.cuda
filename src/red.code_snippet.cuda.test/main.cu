#include "hip/hip_runtime.h"
// includes system
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <stdlib.h>
#include <string>

// includes CUDA
#include "hip/hip_runtime.h"
#include ""

// includes Thrust

// includes project
#include "red_type.h"
#include "red_macro.h"


using namespace std;

__global__
	void kernel_print_vector(int n, const vec_t* v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		printf("v[%4d].x : %20.16lf\n", i, v[i].x);
		printf("v[%4d].y : %20.16lf\n", i, v[i].y);
		printf("v[%4d].z : %20.16lf\n", i, v[i].z);
		printf("v[%4d].w : %20.16lf\n", i, v[i].w);
	}
}

__global__
	void kernel_print_position(int n, const vec_t* r)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < n)
	{
		printf("r[%4d]: %f\n", tid, r[tid]);
		printf("r[%4d].x: %f\n", tid, r[tid].x);
		printf("r[%4d].y: %f\n", tid, r[tid].y);
		printf("r[%4d].z: %f\n", tid, r[tid].z);
		printf("r[%4d].w: %f\n", tid, r[tid].w);
	}
}

void test_print_position(int n, const vec_t* r)
{
	for (int tid = 0; tid < n; tid++)
	{
		printf("r[%4d]: %f\n", tid, r[tid]);
		printf("r[%4d].x: %f\n", tid, r[tid].x);
		printf("r[%4d].y: %f\n", tid, r[tid].y);
		printf("r[%4d].z: %f\n", tid, r[tid].z);
		printf("r[%4d].w: %f\n", tid, r[tid].w);
	}
}

#if 0
int main(int argc, const char** argv)
{
	sim_data_t sim_data;

	sim_data.y.resize(2);
	sim_data.y[0] = new vec_t[8];
	sim_data.y[1] = new vec_t[8];

	sim_data.d_y.resize(2);

	var_t xmax =  1.0;
	var_t xmin = -1.0;
	for (int i = 0; i < 8; i++)
	{
		sim_data.y[0][i].x = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		sim_data.y[0][i].y = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		sim_data.y[0][i].z = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
		sim_data.y[0][i].w = xmin + (var_t)rand() / RAND_MAX * (xmax - xmin);
	}
	test_print_position(8, sim_data.y[0]);

	// Allocate device pointer.
	hipMalloc((void**) &(sim_data.d_y[0]), 8*sizeof(vec_t));
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMalloc failed" << endl;
		return EXIT_FAILURE;
	}
	// Copy pointer content (position and mass) from host to device.
	hipMemcpy(sim_data.d_y[0], sim_data.y[0], 8*sizeof(vec_t), hipMemcpyHostToDevice);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMemcpy failed" << endl;
		return EXIT_FAILURE;
	}
	
	kernel_print_position<<<1, 8>>>(8, sim_data.d_y[0]);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "kernel_print_position failed" << endl;
		return EXIT_FAILURE;
	}
	hipDeviceSynchronize();

	// Allocate pointer.
	vec_t*	v = 0;
	v = (vec_t*)malloc(8 * sizeof(vec_t));
	memset(v, 0, 8 * sizeof(vec_t));

	// Allocate device pointer.
	vec_t*	d_v = 0;
	hipMalloc((void**) &(d_v), 8*sizeof(vec_t));
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMalloc failed" << endl;
		return EXIT_FAILURE;
	}

	// Copy pointer content from host to device.
	hipMemcpy(d_v, v, 8*sizeof(vec_t), hipMemcpyHostToDevice);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "hipMemcpy failed" << endl;
		return EXIT_FAILURE;
	}

	kernel_print_vector<<<1, 8>>>(8, d_v);
	cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus) {
		cerr << "kernel_print_vector failed" << endl;
		return EXIT_FAILURE;
	}
	hipDeviceSynchronize();

	free(v);
	hipFree(d_v);

	delete[] sim_data.y[0];
	delete[] sim_data.y[1];
	hipFree(sim_data.d_y[0]);

	return EXIT_SUCCESS;
}
#endif

int main(int argc, const char** argv)
{
	vector<vector <vec_t*> >	d_f(8);		// size of the outer vector container

	for (int i = 0; i < 8; i++)
	{
		d_f[i].resize(2);					// size of the inner vector container
		for (int j = 0; j < 2; j++)
		{
			d_f[i][j] = new vec_t[4];		// allocate 4 vec_t type element for each i, j pair
		}
	}
}
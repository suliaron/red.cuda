#include <string>

#include "util.h"
#include "red_type.h"
#include "red_macro.h"

void allocate_device_vector(void **d_ptr, size_t size, const char *file, int line)
{
	hipMalloc(d_ptr, size);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw string("hipMalloc failed (allocate_device_vector)");
	}
}

void copy_vector_to_device(void* dst, const void *src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw string("hipMemcpy failed (copy_vector_to_device)");
	}
}

void copy_vector_to_host(void* dst, const void *src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw string("hipMemcpy failed (copy_vector_to_host)");
	}
}

void copy_constant_to_device(const void* dst, const void *src, size_t count)
{
	hipMemcpyToSymbol(HIP_SYMBOL(dst), src, count);
	hipError_t cudaStatus = HANDLE_ERROR(hipGetLastError());
	if (hipSuccess != cudaStatus)
	{
		throw string("hipMemcpyToSymbol failed (copy_constant_to_device)");
	}
}

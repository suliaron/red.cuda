#include <iostream>
#include <iomanip>
#include <fstream>

#include "tbp1D.h"

#include "redutilcu.h"
#include "red_constants.h"

using namespace std;
using namespace redutilcu;


tbp1D::tbp1D(uint16_t n_ppo, computing_device_t comp_dev) :
	ode(1, 2, n_ppo, 1, comp_dev)
{
	initialize();
	allocate_storage();
}

tbp1D::~tbp1D()
{
	deallocate_storage();
}

void tbp1D::initialize()
{
	h_md    = 0x0;
	h_epoch = 0x0;

	h       = 0.0;
}

void tbp1D::allocate_storage()
{
	allocate_host_storage();
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		allocate_device_storage();
	}
}

void tbp1D::allocate_host_storage()
{
	ALLOCATE_HOST_VECTOR((void**)&(h_md),    n_obj * sizeof(tbp1D_t::metadata_t));
	ALLOCATE_HOST_VECTOR((void**)&(h_epoch), n_obj * sizeof(var_t));
}

void tbp1D::allocate_device_storage()
{
	ALLOCATE_DEVICE_VECTOR((void**)&(d_md),    n_obj * sizeof(tbp1D_t::metadata_t));
	ALLOCATE_DEVICE_VECTOR((void**)&(d_epoch), n_obj * sizeof(var_t));
}

void tbp1D::deallocate_storage()
{
	deallocate_host_storage();
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		deallocate_device_storage();
	}
}

void tbp1D::deallocate_host_storage()
{
	FREE_HOST_VECTOR((void **)&(h_md));
	FREE_HOST_VECTOR((void **)&(h_epoch));
}

void tbp1D::deallocate_device_storage()
{
	FREE_DEVICE_VECTOR((void **)&(h_md));
	FREE_DEVICE_VECTOR((void **)&(h_epoch));
}

void tbp1D::calc_dy(uint16_t stage, ttt_t curr_t, const var_t* y_temp, var_t* dy)
{
	if (COMPUTING_DEVICE_CPU == comp_dev)
	{
		cpu_calc_dy(stage, curr_t, y_temp, dy);
	}
	else
	{
		gpu_calc_dy(stage, curr_t, y_temp, dy);
	}
}

void tbp1D::calc_energy()
{
	const tbp1D_t::param_t* p = (tbp1D_t::param_t*)h_p;

	h = 0.5 * SQR(h_y[1]) - p[0].mu / h_y[0];
}

void tbp1D::cpu_calc_dy(uint16_t stage, ttt_t curr_t, const var_t* y_temp, var_t* dy)
{
	const tbp1D_t::param_t* p = (tbp1D_t::param_t*)h_p;

	dy[0] = y_temp[1];                    // dx1 / dt = x2
	dy[1] = -p[0].mu / SQR(y_temp[0]);    // dx2 / dt = -mu / (x1*x1)
}

void tbp1D::gpu_calc_dy(uint16_t stage, ttt_t curr_t, const var_t* y_temp, var_t* dy)
{
	throw string("The gpu_calc_dy() is not implemented.");
}

void tbp1D::load(string& path)
{
	ifstream input;

	cout << "Loading " << path << " ";

	data_rep_t repres = (file::get_extension(path) == "txt" ? DATA_REPRESENTATION_ASCII : DATA_REPRESENTATION_BINARY);
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		input.open(path.c_str());
		if (input) 
		{
			load_ascii(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	case DATA_REPRESENTATION_BINARY:
		input.open(path.c_str(), ios::binary);
		if (input) 
		{
			load_binary(input);
		}
		else 
		{
			throw string("Cannot open " + path + ".");
		}
		break;
	}
	input.close();

	cout << " done" << endl;
}

void tbp1D::load_ascii(ifstream& input)
{
	tbp1D_t::param_t* p = (tbp1D_t::param_t*)h_p;

	for (uint32_t i = 0; i < n_obj; i++)
	{
		load_ascii_record(input, &h_epoch[i], &h_md[i], &p[i], &h_y[i], &h_y[i+1]);
	}
}

void tbp1D::load_ascii_record(ifstream& input, ttt_t* t, tbp1D_t::metadata_t *md, tbp1D_t::param_t* p, var_t* x, var_t* vx)
{
	string name;

	// epoch
	input >> *t;
	// name
	input >> name;
	if (name.length() > 30)
	{
		name = name.substr(0, 30);
	}
	obj_names.push_back(name);
	// id
	input >> md->id;
	// mu = k^2*(m1 + m2)
	input >> p->mu;
	// position
	input >> *x;
	// velocity
	input >> *vx;
}

void tbp1D::load_binary(ifstream& input)
{
	throw string("The load_binary() is not implemented.");
}

void tbp1D::print_result(ofstream& sout, data_rep_t repres)
{
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		print_result_ascii(sout);
		break;
	case DATA_REPRESENTATION_BINARY:
		print_result_binary(sout);
		break;
	}
}

void tbp1D::print_result_ascii(ofstream& sout)
{
	static uint32_t int_t_w  =  8;
	static uint32_t var_t_w  = 25;

	sout.precision(16);
	sout.setf(ios::right);
	sout.setf(ios::scientific);

	for (uint32_t i = 0; i < n_obj; i++)
    {
		uint32_t orig_idx = h_md[i].id - 1;

		sout << setw(var_t_w) << t << SEP                       /* time of the record [day] (double)           */
			 << setw(     30) << obj_names[orig_idx] << SEP     /* name of the body         (string = 30 char) */ 
		// Print the metadata for each object
        << setw(int_t_w) << h_md[i].id << SEP;

		// Print the parameters for each object
		for (uint16_t j = 0; j < n_ppo; j++)
		{
			uint32_t param_idx = i * n_ppo + j;
			sout << setw(var_t_w) << h_p[param_idx] << SEP;
		}
		// Print the variables for each object
		for (uint16_t j = 0; j < n_vpo; j++)
		{
			uint32_t var_idx = i * n_vpo + j;
			sout << setw(var_t_w) << h_y[var_idx];
			if (j < n_vpo - 1)
			{
				sout << SEP;
			}
			else
			{
				sout << endl;
			}
		}
	}
	sout.flush();
}

void tbp1D::print_result_binary(ofstream& sout)
{
	throw string("The print_result_binary() is not implemented.");
}

void tbp1D::print_integral_data(ofstream& sout, data_rep_t repres)
{
	switch (repres)
	{
	case DATA_REPRESENTATION_ASCII:
		print_integral_data_ascii(sout);
		break;
	case DATA_REPRESENTATION_BINARY:
		print_integral_data_binary(sout);
		break;
	}
}

void tbp1D::print_integral_data_ascii(ofstream& sout)
{
	static uint32_t int_t_w  =  8;
	static uint32_t var_t_w  = 25;

	sout.precision(16);
	sout.setf(ios::right);
	sout.setf(ios::scientific);

	sout << setw(var_t_w) << t << SEP                       /* time of the record [day] (double)           */
		 << h << endl;

	sout. flush();
}

void tbp1D::print_integral_data_binary(ofstream& sout)
{
	throw string("The print_integral_data_binary() is not implemented.");
}


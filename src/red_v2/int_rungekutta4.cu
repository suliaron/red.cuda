#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ode.h"
#include "int_rungekutta4.h"

#include "red_macro.h"
#include "redutilcu.h"

#define	LAMBDA	1.0/10.0

// The Runge-Kutta matrix
var_t int_rungekutta4::a[] = 
{ 
	1.0/2.0,
	0.0,     1.0/2.0,
	0.0,     0.0,     1.0,
    1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0
};
// weights
var_t int_rungekutta4::b[]  = { 1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0 - LAMBDA, LAMBDA };
var_t int_rungekutta4::bh[] = { 1.0/6.0, 1.0/3.0, 1.0/3.0, 1.0/6.0         ,    0.0 };
// nodes
ttt_t int_rungekutta4::c[]  = {     0.0, 1.0/2.0, 1.0/2.0, 1.0, 1.0                 };
// The starting index of the RK matrix for the stages
uint16_t int_rungekutta4::a_idx[] = {0, 1, 3, 6};

namespace rk4_kernel
{
// a_i = b_i + F * c_i
static __global__
	void sum_vector(var_t* a, const var_t* b, var_t F, const var_t* c, uint32_t n)
{
	uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;

	while (n > tid)
	{
		a[tid] = b[tid] + F * c[tid];
		tid += stride;
	}
}
} /* namespace rk4_kernel */

int_rungekutta4::int_rungekutta4(ode& f, ttt_t dt, bool adaptive, var_t tolerance, computing_device_t comp_dev) :
	integrator(f, dt, adaptive, tolerance, (adaptive ? 5 : 4), comp_dev)
{
	name    = "Runge-Kutta4";
	n_order = 4;
}

int_rungekutta4::~int_rungekutta4()
{}

void int_rungekutta4::calc_lin_comb(var_t* y, const var_t* y_n, const var_t* coeff, uint16_t n_coeff, uint32_t n_var)
{
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		// rk4_kernel::calc_lin_comb
		CUDA_CHECK_ERROR();
	}
	else
	{
		cpu_calc_lin_comb(y, y_n, coeff, n_coeff, n_var);
	}
}

//void int_rungekutta4::calc_ytemp(uint16_t stage)
//{
//	if (COMPUTING_DEVICE_GPU == comp_dev)
//	{
//		// rk4_kernel::calc_ytemp
//		CUDA_CHECK_ERROR();
//	}
//	else
//	{
//		cpu_calc_ytemp(stage);
//	}
//}

void int_rungekutta4::calc_error(uint32_t n)
{
	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		// rk4_kernel::calc_error
		CUDA_CHECK_ERROR();
	}
	else
	{
		cpu_calc_error(n);
	}
}

//void int_rungekutta4::calc_y_np1()
//{
//	if (COMPUTING_DEVICE_GPU == comp_dev)
//	{
//		// rk4_kernel::calc_y_np1
//		CUDA_CHECK_ERROR();
//	}
//	else
//	{
//		cpu_calc_y_np1();
//	}
//}

// a_i = b_i + F * c_i
//void int_rungekutta4::cpu_sum_vector(var_t* a, const var_t* b, var_t F, const var_t* c, uint32_t n)
//{
//	for (uint32_t tid = 0; tid < n; tid++)
//	{
//		a[tid] = b[tid] + F * c[tid];
//	}
//}

void int_rungekutta4::cpu_calc_lin_comb(var_t* y, const var_t* y_n, const var_t* coeff, uint16_t n_coeff, uint32_t n_var)
{
	for (uint32_t i = 0; i < n_var; i++)
	{
		var_t dy = 0.0;
		for (uint16_t j = 0; j < n_coeff; j++)
		{
			if (0.0 == coeff[j])
			{
				continue;
			}
			dy += coeff[j] * h_k[j][i];
		}
		y[i] = y_n[i] + dy;
	}
}

//void int_rungekutta4::cpu_calc_ytemp(uint16_t stage)
//{
//	for (uint32_t i = 0; i < f.n_var; i++)
//	{
//		var_t dy = 0.0;
//		for (uint16_t j = 0; j < stage; j++)
//		{
//			if (0.0 == a[a_idx[stage-1] + j])
//			{
//				continue;
//			}
//			dy += a[a_idx[stage-1] + j] * h_k[j][i];
//		}
//
//		h_ytemp[i] = f.h_y[i] + dt_try * dy;
//	}
//}
//
//void int_rungekutta4::cpu_calc_y_np1()
//{
//	for (uint32_t i = 0; i < f.n_var; i++)
//	{
//		var_t dy = 0.0;
//		for (uint16_t j = 0; j < n_order; j++)
//		{
//			if (0.0 == b[j])
//			{
//				continue;
//			}
//			dy += b[j] * h_k[j][i];
//		}
//		f.h_yout[i] = f.h_y[i] + dt_try * dy;
//	}
//}

void int_rungekutta4::cpu_calc_error(uint32_t n)
{
	for (uint32_t i = 0; i < n; i++)
	{
		h_err[i] = fabs(h_k[3][i] - h_k[4][i]);
	}
}

ttt_t int_rungekutta4::step()
{
	static string err_msg1 = "The integrator could not provide the approximation of the solution with the specified tolerance.";

	static const uint16_t n_a = sizeof(int_rungekutta4::a) / sizeof(int_rungekutta4::a[0]);
	static const uint16_t n_b = sizeof(int_rungekutta4::b) / sizeof(int_rungekutta4::b[0]);
	static var_t aa[n_a];
	static var_t bb[n_b];

	if (COMPUTING_DEVICE_GPU == comp_dev)
	{
		redutilcu::set_kernel_launch_param(f.n_var, THREADS_PER_BLOCK, grid, block);
	}

	uint16_t stage = 0;
	t = f.t;
	// Calculate initial differentials and store them into h_k
	f.calc_dy(stage, t, f.h_y, h_k[stage]);

	var_t max_err = 0.0;
	uint16_t iter = 0;
	do
	{
		dt_did = dt_try;
		// Compute in advance the dt_try * coefficients to save n_var multiplication per stage
		for (uint16_t i = 0; i < n_a; i++)
		{
			aa[i] = dt_try * a[i];
		}
		for (uint16_t i = 0; i < n_b; i++)
		{
			bb[i] = dt_try * bh[i];
		}

		for (stage = 1; stage < 4; stage++)
		{
			t = f.t + c[stage] * dt_try;
			cpu_calc_lin_comb(h_ytemp, f.h_y, &aa[a_idx[stage-1]], stage, f.n_var);
			//calc_ytemp(stage);
			f.calc_dy(stage, t, h_ytemp, h_k[stage]);
		}
		// y_(n+1) = yn + dt*(1/6*k1 + 1/3*k2 + 1/3*k3 + 1/6*k4) + O(dt^5)
		// So far we have stage (=4) number of k vectors
		cpu_calc_lin_comb(f.h_yout, f.h_y, bb, stage, f.n_var);
		//calc_y_np1();

		if (adaptive)
		{
			// Here stage = 4
			t = f.t + c[stage] * dt_try;
			f.calc_dy(stage, t, f.h_yout, h_k[stage]);

			// calculate: err = abs(k4 - k5)
			calc_error(f.n_var);
			max_err = dt_try * LAMBDA * get_max_error(f.n_var);
			dt_try *= 0.9 * pow(tolerance / max_err, 1.0/(n_order));
		}
		iter++;
	} while (adaptive && max_iter > iter && dt_min < dt_try && max_err > tolerance);

	if (max_iter <= iter)
	{
		throw string(err_msg1 + " The number of iteration exceeded the limit.");
	}
	if (dt_min >= dt_try)
	{
		throw string(err_msg1 + " The stepsize is smaller than the limit.");
	}

	update_counters(iter);

	t = f.t + dt_did;
	f.tout = t;
	f.swap();

	return dt_did;
}

#undef LAMBDA
